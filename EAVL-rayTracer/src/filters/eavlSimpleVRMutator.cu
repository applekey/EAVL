#include "hip/hip_runtime.h"
#include "eavlException.h"
#include "eavlExecutor.h"
#include "eavlSimpleVRMutator.h"
#include "eavlMapOp.h"
#include "eavlColor.h"
#include "eavlPrefixSumOp_1.h"
#include "eavlReduceOp_1.h"
#include "eavlGatherOp.h"
#include "eavlSimpleReverseIndexOp.h"
#include "eavlRayExecutionMode.h"
#include "eavlRTUtil.h"
#ifdef HAVE_CUDA
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#endif

#define COLOR_MAP_SIZE 1024

long int scounter = 0;
long int skipped = 0;

texture<float4> scalars_tref;
texture<float4> cmap_tref;

eavlConstTexArray<float4>* color_map_array;
eavlConstTexArray<float4>* scalars_array;

#define PASS_ESTIMATE_FACTOR  2.5f

//-------------------------------------------------

eavlSimpleVRMutator::eavlSimpleVRMutator()
{   
    cpu = eavlRayExecutionMode::isCPUOnly();


    opacityFactor = 1.f;
    height = 100;
    width  = 100;    
    setNumPasses(1); //default number of passes
    samples                = NULL;
    framebuffer            = NULL;
    zBuffer                = NULL;
    minSample              = NULL;
    iterator               = NULL;
    screenIterator         = NULL;
    colormap_raw           = NULL;
    minPasses              = NULL;
    maxPasses              = NULL;
    currentPassMembers     = NULL;
    passNumDirty           = true;
    indexScan              = NULL;
    reverseIndex           = NULL;
    scalars_array          = NULL; 

    ir = new eavlArrayIndexer(4,0);
    ig = new eavlArrayIndexer(4,1);
    ib = new eavlArrayIndexer(4,2);
    ia = new eavlArrayIndexer(4,3);
    ssa    = NULL;
    ssb    = NULL;
    ssc    = NULL;
    ssd    = NULL;
    tetSOA = NULL;
    mask   = NULL;
    rgba   = NULL;
    scene = new eavlVRScene();

    geomDirty = true;
    sizeDirty = true;

    numTets = 0;
    nSamples = 500;
    passCount = new eavlIntArray("",1,1); 
    i1 = new eavlArrayIndexer(3,0);
    i2 = new eavlArrayIndexer(3,1);
    i3 = new eavlArrayIndexer(3,2);
    idummy = new eavlArrayIndexer();
    idummy->mod = 1 ;
    dummy = new eavlFloatArray("",1,2);

    verbose = false;

    setDefaultColorMap(); 
    isTransparentBG = false;
    
    //
    // Init sample buffer
    // 
    dx = width;
    dy = height;
    dz = nSamples;
    xmin = 0;
    ymin = 0;
    zmin = 0;  
}

//-------------------------------------------------

eavlSimpleVRMutator::~eavlSimpleVRMutator()
{
    if(verbose) cout<<"Destructor"<<endl;
    deleteClassPtr(samples);
    deleteClassPtr(framebuffer);
    deleteClassPtr(zBuffer);
    deleteClassPtr(minSample);
    deleteClassPtr(rgba);
    deleteClassPtr(scene);
    deleteClassPtr(ssa);
    deleteClassPtr(ssb);
    deleteClassPtr(ssc);
    deleteClassPtr(ssd);
    deleteClassPtr(iterator);
    deleteClassPtr(i1);
    deleteClassPtr(i2);
    deleteClassPtr(i3);
    deleteClassPtr(ir);
    deleteClassPtr(ig);
    deleteClassPtr(ib);
    deleteClassPtr(ia);
    deleteClassPtr(idummy);
    deleteClassPtr(minPasses);
    deleteClassPtr(maxPasses);
    deleteClassPtr(indexScan);
    deleteClassPtr(mask);
    deleteClassPtr(dummy);
    deleteClassPtr(currentPassMembers);
    deleteClassPtr(reverseIndex);
    deleteClassPtr(screenIterator);

    freeTextures();
    freeRaw();

}

//-------------------------------------------------

void eavlSimpleVRMutator::getBBoxPixelExtent(eavlPoint3 &smins, eavlPoint3 &smaxs)
{
    float xmin = FLT_MAX;
    float xmax = -FLT_MAX;
    float ymin = FLT_MAX;
    float ymax = -FLT_MAX;
    float zmin = FLT_MAX;
    float zmax = -FLT_MAX;

    eavlPoint3 bbox[2];
    bbox[0] = smins;
    bbox[1] = smaxs;
    for(int x = 0; x < 2 ; x++)
    {
        for(int y = 0; y < 2 ; y++)
        {
            for(int z = 0; z < 2 ; z++)
            {
                eavlPoint3 temp(bbox[x].x, bbox[y].y, bbox[z].z);

                eavlPoint3 t = view.P * view.V * temp;
                t.x = (t.x*.5+.5)  * view.w;
                t.y = (t.y*.5+.5)  * view.h;
                t.z = (t.z*.5+.5)  * (float) nSamples;
                zmin = min(zmin,t.z);
                ymin = min(ymin,t.y);
                xmin = min(xmin,t.x);
                zmax = max(zmax,t.z);
                ymax = max(ymax,t.y);
                xmax = max(xmax,t.x);
            }
        }
    }
  
  
    xmin-=.001f;
    xmax+=.001f;
    ymin-=.001f;
    ymax+=.001f;
    zmin+=.001f;
    xmin = floor(fminf(fmaxf(0.f, xmin),view.w));
    xmax = ceil(fminf(fmaxf(0.f, xmax),view.w));
    ymin = floor(fminf(fmaxf(0.f, ymin),view.h));
    ymax = ceil(fminf(fmaxf(0.f, ymax),view.h));
    zmin = floor(fminf(fmaxf(0.f, zmin),nSamples));
    zmax = ceil(fminf(fmaxf(0.f, zmax),nSamples));
    smins.x = xmin;
    smins.y = ymin;
    smins.z = zmin;

    smaxs.x = xmax;
    smaxs.y = ymax;
    smaxs.z = zmax;
    //cout<<"BBOX "<<smins<<smaxs<<endl;
}

//-------------------------------------------------

//
//  TODO: This is no longer technically screen space. All coordinates
//        are offset into "sample space" which is a subset of screen space.
//
struct ScreenSpaceFunctor
{   
    float4 *xverts;
    float4 *yverts; 
    float4 *zverts;
    eavlView         view;
    int              nSamples;
    float            xmin;
    float            ymin;
    float            zmin;
    ScreenSpaceFunctor(float4 *_xverts, float4 *_yverts,float4 *_zverts, eavlView _view, int _nSamples, int _xmin, int _ymin, int _zmin)
    : view(_view), xverts(_xverts),yverts(_yverts),zverts(_zverts), nSamples(_nSamples), xmin(_xmin), ymin(_ymin), zmin(_zmin)
    {}

    EAVL_FUNCTOR tuple<float,float,float,float,float,float,float,float,float,float,float,float> operator()(tuple<int> iterator)
    {
        int tet = get<0>(iterator);
        eavlPoint3 mine(FLT_MAX,FLT_MAX,FLT_MAX);
        eavlPoint3 maxe(-FLT_MAX,-FLT_MAX,-FLT_MAX);

        float* v[3];
        v[0] = (float*)&xverts[tet]; //x
        v[1] = (float*)&yverts[tet]; //y
        v[2] = (float*)&zverts[tet]; //z

        eavlPoint3 p[4];
        //int clipped = 0;
        for( int i=0; i< 4; i++)
        {   
            p[i].x = v[0][i];
            p[i].y = v[1][i]; 
            p[i].z = v[2][i];

            eavlPoint3 t = view.P * view.V * p[i];
            //cout<<"Before"<<t<<endl;
            // if(t.x > 1 || t.x < -1) clipped = 1;
            // if(t.y > 1 || t.y < -1) clipped = 1;
            // if(t.z > 1 || t.z < -1) clipped = 1;
            p[i].x = (t.x*.5+.5)  * view.w -xmin;
            p[i].y = (t.y*.5+.5)  * view.h - ymin;
            p[i].z = (t.z*.5+.5)  * (float) nSamples -zmin;
            //cout<<"After "<<p[i]<<endl;
        }
        

        return tuple<float,float,float,float,float,float,float,float,float,float,float,float>(p[0].x, p[0].y, p[0].z,
                                                                                                  p[1].x, p[1].y, p[1].z,
                                                                                                  p[2].x, p[2].y, p[2].z,
                                                                                                  p[3].x, p[3].y, p[3].z);
    }

   

};

//-------------------------------------------------

struct PassRange
{   
    float4 *xverts;
    float4 *yverts;
    float4 *zverts;
    eavlView         view;
    int              nSamples; // this is now the number of samples that the inside image space
    float            mindepth;
    float            maxdepth;
    int              numPasses;
    int              passStride;
    int              CellThreshold;
    float            zmin;// need this to transate into "sample space"
    int              dz;
    int              mysampleLCFlag;

    PassRange(float4 *_xverts, float4 *_yverts,float4 *_zverts, eavlView _view, int _nSamples, int _numPasses, int _zmin, int _dz, int _sampleLCFlag)
    : view(_view), xverts(_xverts),yverts(_yverts),zverts(_zverts), nSamples(_nSamples), numPasses(_numPasses), zmin(_zmin), dz(_dz), mysampleLCFlag(_sampleLCFlag)
    {
        CellThreshold = 100;
        passStride = dz / numPasses;
        //if it is not evenly divided add one pixel row so we cover all pixels
        if(((int)nSamples % numPasses) != 0) passStride++;
        
    }

    EAVL_FUNCTOR tuple<byte,byte,int> operator()(tuple<int> iterator)
    {
        int tet = get<0>(iterator);
        eavlPoint3 mine(FLT_MAX,FLT_MAX,FLT_MAX);
        eavlPoint3 maxe(-FLT_MAX,-FLT_MAX,-FLT_MAX);
        float* v[3];
        v[0] = (float*)&xverts[tet]; //x
        v[1] = (float*)&yverts[tet]; //y
        v[2] = (float*)&zverts[tet]; //z

        int clipped = 0;
        eavlPoint3 p[4];

        for( int i=0; i < 4; i++)
        {   
            p[i].x = v[0][i];
            p[i].y = v[1][i]; 
            p[i].z = v[2][i];

            eavlPoint3 t = view.P * view.V * p[i];
            if(t.x > 1 || t.x < -1) clipped = 1;
            if(t.y > 1 || t.y < -1) clipped = 1;
            if(t.z > 1 || t.z < -1) clipped = 1;
            p[i].x = (t.x*.5+.5)  * view.w;
            p[i].y = (t.y*.5+.5)  * view.h;
            p[i].z = (t.z*.5+.5)  * (float) nSamples - zmin; //into sample space

        }
        //Looping over faces
        for(int i=0; i<4; i++)
        {    
            //looping over dimenstions
            for (int d=0; d<3; ++d)
            {
                    mine[d] = min(p[i][d], mine[d] );
                    maxe[d] = max(p[i][d], maxe[d] );
            }
        }
        //if the tet stradles the edge, dump it TODO: extra check to make sure it is all the way outside
        float mn = min(mine[2],min(mine[1],min(mine[0], float(1e9) )));
        if(mn < 0) clipped = 1;
        
        if(clipped == 1) return tuple<byte,byte,int>(255,255,0); //not part of any pass
        int minPass = 0;
        int maxPass = 0;
        // now transate into sample space
        minPass = mine[2] / passStride; //min z coord
        maxPass = maxe[2] / passStride; //max z coord

        int tetNumofSample = ((maxe[0] - mine[0]) * (maxe[0] - mine[0])) + ((maxe[1] - mine[1]) * (maxe[1] - mine[1])) + ((maxe[2] - mine[2]) * (maxe[2] - mine[2]));

    if(mysampleLCFlag == 0)
    { 
      if(tetNumofSample > CellThreshold)
       return tuple<byte,byte,int>(255,255,tetNumofSample);
      
       else
        return tuple<byte,byte,int>(minPass, maxPass,0);
    }// if sampleLCFlag == 0 which mean only sample small cells
    else
    {
        return tuple<byte,byte,int>(minPass, maxPass,0);
    }// sampleLCFlag == 1 which means sample large cells
    
    }
};

//-------------------------------------------------

float EAVL_HOSTDEVICE ffmin(const float &a, const float &b)
{
    #if __CUDA_ARCH__
        return fmin(a,b);
    #else
        return (a > b) ? b : a;
    #endif
}

//-------------------------------------------------

float EAVL_HOSTDEVICE ffmax(const float &a, const float &b)
{
     #if __CUDA_ARCH__
        return fmax(a,b);
    #else
        return (a > b) ? a : b;
    #endif
    
}

//-------------------------------------------------

//
// Incoming coordinate are in sample space
//
struct SampleFunctor3
{   
    const eavlConstTexArray<float4> *scalars;
    eavlView         view;
    int              nSamples;
    float*           samples;
    float*           fb;
    int              passMinZPixel;
    int              passMaxZPixel;
    int              zSize;
    int              dx;
    int              dy;
    int              dz;
    int              minx;
    int              miny;
    SampleFunctor3(const eavlConstTexArray<float4> *_scalars, eavlView _view, int _nSamples, float* _samples, int _passMinZPixel, int _passMaxZPixel,int numZperPass, float* _fb, int _dx, int _dy, int _dz, int _minx, int _miny)
    : view(_view), scalars(_scalars), nSamples(_nSamples), samples(_samples), dx(_dx), dy(_dy), dz(_dz), minx(_minx), miny(_miny)
    {
        
        passMaxZPixel  = min(int(dz-1), _passMaxZPixel);
        passMinZPixel  = max(0, _passMinZPixel);
        zSize = numZperPass;
        fb = _fb;
        //printf("Min and max z pixel : %d %d \n", passMinZPixel, passMaxZPixel);
    }

    EAVL_FUNCTOR tuple<float> operator()(tuple<int,float,float,float,float,float,float,float,float,float,float,float,float> inputs )
    {
        int tet = get<0>(inputs);
        
        eavlVector3 p[4]; //TODO vectorize
        p[0].x = get<1>(inputs);
        p[0].y = get<2>(inputs);
        p[0].z = get<3>(inputs);

        p[1].x = get<4>(inputs);
        p[1].y = get<5>(inputs);
        p[1].z = get<6>(inputs);

        p[2].x = get<7>(inputs);
        p[2].y = get<8>(inputs);
        p[2].z = get<9>(inputs);

        p[3].x = get<10>(inputs);
        p[3].y = get<11>(inputs);
        p[3].z = get<12>(inputs);

        eavlVector3 v[3];
        for(int i = 1; i < 4; i++)
        {
            v[i-1] = p[i] - p[0];
        }

        //                  a         b            c       d
        //float d1 = D22(mat[1][1], mat[1][2], mat[2][1], mat[2][2]);
        float d1 = v[1].y * v[2].z - v[2].y * v[1].z;
        //float d2 = D22(mat[1][0], mat[1][2], mat[2][0], mat[2][2]);
        float d2 = v[0].y * v[2].z - v[2].y *  v[0].z;
        //float d3 = D22(mat[1][0], mat[1][1], mat[2][0], mat[2][1]);
        float d3 = v[0].y * v[1].z - v[1].y * v[0].z;

        float det = v[0].x * d1 - v[1].x * d2 + v[2].x * d3;

        if(det == 0) return tuple<float>(0.f); // dirty degenerate tetrahedron
        det  = 1.f  / det;

        //D22(mat[0][1], mat[0][2], mat[2][1], mat[2][2]);
        float d4 = v[1].x * v[2].z - v[2].x * v[1].z;
        //D22(mat[0][1], mat[0][2], mat[1][1], mat[1][2])
        float d5 = v[1].x * v[2].y - v[2].x * v[1].y;
        //D22(mat[0][0], mat[0][2], mat[2][0], mat[2][2]) 
        float d6 = v[0].x * v[2].z- v[2].x * v[0].z; 
        //D22(mat[0][0], mat[0][2], mat[1][0], mat[1][2])
        float d7 = v[0].x * v[2].y - v[2].x * v[0].y;
        //D22(mat[0][0], mat[0][1], mat[2][0], mat[2][1])
        float d8 = v[0].x * v[1].z - v[1].x * v[0].z;
        //D22(mat[0][0], mat[0][1], mat[1][0], mat[1][1])
        float d9 = v[0].x * v[1].y - v[1].x * v[0].y;
        /* need the extents again, just recalc */
        eavlPoint3 mine(FLT_MAX,FLT_MAX,FLT_MAX);
        eavlPoint3 maxe(-FLT_MAX,-FLT_MAX,-FLT_MAX);
       
        for(int i=0; i<4; i++)  //these two loops cost 2 registers
        {    
            for (int d=0; d<3; ++d) 
            {
                    mine[d] = min(p[i][d], mine[d] );
                    maxe[d] = max(p[i][d], maxe[d] );
            }
        } 

        // for(int i = 0; i < 3; i++) mine[i] = max(mine[i],0.f);
        // /*clamp*/
        maxe[0] = min(float(dx-1.f), maxe[0]); //??  //these lines cost 14 registers
        maxe[1] = min(float(dy - 1.f), maxe[1]);
        maxe[2] = min(float(passMaxZPixel), maxe[2]);
        mine[2] = max(float(passMinZPixel), mine[2]);
        //cout<<p[0]<<p[1]<<p[2]<<p[3]<<endl;
        int xmin = ceil(mine[0]);
        int xmax = floor(maxe[0]);
        int ymin = ceil(mine[1]);
        int ymax = floor(maxe[1]);
        int zmin = ceil(mine[2]);
        int zmax = floor(maxe[2]);

        float4 s = scalars->getValue(scalars_tref, tet);
        //cerr<<" X "<<xmin<<" to "<<xmax<<"\n";
        //cerr<<" Y "<<ymin<<" to "<<ymax<<"\n";
        for(int x = xmin; x <= xmax; ++x)
        {
            for(int y = ymin; y <= ymax; ++y)
            { 
                int pixel = ( (y+miny) * view.w + x + minx);
                if(fb[pixel * 4 + 3] >= 1) {continue;} //TODO turn this on using sample space to screen space
                
                int startindex = (y * dx + x) * zSize;//dx*(y + dy*(z -passMinZPixel));
                #pragma ivdep
                for(int z=zmin; z<=zmax; ++z)
                {

                    float w1 = x - p[0].x; 
                    float w2 = y - p[0].y; 
                    float w3 = z - p[0].z; 

                    float xx =   w1 * d1 - w2 * d4 + w3 * d5;
                    xx *= det; 

                    float yy = - w1 * d2 + w2 * d6 - w3 * d7; 
                    yy *= det;

                    float zz =   w1 * d3 - w2 * d8 + w3 * d9;
                    zz *= det;
                    w1 = xx; 
                    w2 = yy; 
                    w3 = zz; 

                    float w0 = 1.f - w1 - w2 - w3;

                    int index3d = startindex + z;
                    float lerped = w0*s.x + w1*s.y + w2*s.z + w3*s.w;
                    float a = ffmin(w0,ffmin(w1,ffmin(w2,w3)));
                    float b = ffmax(w0,ffmax(w1,ffmax(w2,w3)));
                    if((a >= 0.f && b <= 1.f)) 
                    {
                        samples[index3d] = lerped;
                       if(x == 359 && y == 282)
                        cerr<<"Cell "<<tet<<"\n";
                        //cerr<<"Z "<<z<<" value "<<samples[index3d]<<"\n";
                      // cerr<<"HEEEEEELLO\n";
                        //if(lerped < 0 || lerped >1) printf("Bad lerp %f ",lerped);
                    }
                     
                   

                }//z
            }//y                                                                                                                                                                                           
        }//x

        return tuple<float>(0.f);
    }
};

//-------------------------------------------------

struct CompositeFunctorFB
{   
    const eavlConstTexArray<float4> *colorMap;
    eavlView         view;
    int              nSamples;
    float*           samples;
    int              h;
    int              w;
    int              ncolors;
    float            mindepth;
    float            maxdepth;
    eavlPoint3       minComposite;
    eavlPoint3       maxComposite;
    int              zOffest;
    bool             finalPass;
    int              maxSIndx;
    int              minZPixel;

    int              dx;
    int              dy;
    //int              dz;
    int              xmin;
    int              ymin;

    CompositeFunctorFB( eavlView _view, int _nSamples, float* _samples, const eavlConstTexArray<float4> *_colorMap, int _ncolors, eavlPoint3 _minComposite, eavlPoint3 _maxComposite, int _zOffset, bool _finalPass, int _maxSIndx, int _minZPixel, int _dx, int _dy, int _xmin, int _ymin)
    : view(_view), nSamples(_nSamples), samples(_samples), colorMap(_colorMap), ncolors(_ncolors), minComposite(_minComposite), maxComposite(_maxComposite), finalPass(_finalPass), maxSIndx(_maxSIndx),
      dx(_dx), dy(_dy), xmin(_xmin), ymin(_ymin)
    {
        w = view.w;
        h = view.h;
        zOffest = _zOffset;
        minZPixel = _minZPixel;
    }
 
    EAVL_FUNCTOR tuple<float,float,float,float,int> operator()(tuple<int, float, float, float, float, int> inputs )
    {
        int idx = get<0>(inputs);
        int x = idx%w;
        int y = idx/w;
        int minZsample = get<5>(inputs);
        //get the incoming color and return if the opacity is already 100%
        float4 color= {get<1>(inputs),get<2>(inputs),get<3>(inputs),get<4>(inputs)};
        if(color.w >= 1) return tuple<float,float,float,float,int>(color.x, color.y, color.z,color.w, minZsample);
        //cerr<<"Before \n";
        x-= xmin;
        y-= ymin;
        //pixel outside the AABB of the data set
        if((x >= dx) || (x < 0) || ( y >= dy) || (y < 0))
        {
            return tuple<float,float,float,float,int>(0.f,0.f,0.f,0.f, minZsample);
        }
        //cerr<<"After is \n";
        for(int z = 0 ; z < zOffest; z++)
        {
                //(x + view.w*(y + zSize*z));
            int index3d = (y*dx + x)*zOffest + z;//(x + dx*(y + dy*(z))) ;//
            
            //printf("Coord = (%f,%f,%f) %d ",x,y,z, index3d);
            float value =  samples[index3d];//tsamples->getValue(samples_tref, index3d);// samples[index3d];
            
            //takes init value -1 if it was a large cell 
            if (value <= 0.f || value > 1.f)
                continue; //cerr<<"Value "<<value<<"\n";
        
            int colorindex = float(ncolors-1) * value;
            float4 c = colorMap->getValue(cmap_tref, colorindex);
            //cout<<"color for value "<<value<<" is "<<color.x<<" "<<color.y<<" "<<color.z<<" "<<color.w<<"\n";
            c.w *= (1.f - color.w); 
            color.x = color.x  + c.x * c.w;
            color.y = color.y  + c.y * c.w;
            color.z = color.z  + c.z * c.w;
            color.w = c.w + color.w;

                  minZsample = min(minZsample, minZPixel + z); //we need the closest sample to get depth buffer 
            if(color.w >=1 ) break;

        }
   	
//	cerr<<"Min Sample "<<minZsample<<"\n"; 
        return tuple<float,float,float,float,int>(min(1.f, color.x),  min(1.f, color.y),min(1.f, color.z),min(1.f,color.w), minZsample);
        
    }
   

};
//-------------------------------------------------
/*
struct PartialComposite
{

public:
    int startIndex;
    int endIndex;
    int x;
    int y;
    float4 color;

};
/*
struct PixelPartials
{
public:
    int numOfPartials;
    PartialComposite* myPartialsArray;

};*/
//typedef eavlConcreteArray<PartialComposite> eavlPartialComp;
//-------------------------------------------------

struct TestMyStruct
{
    int factor;
    float* ray;
    eavlIntArray* offesetPartials;

    TestMyStruct(float* _rays,eavlIntArray* _offesetPartials):ray(_rays), offesetPartials(_offesetPartials)
    {
        factor = 2;
    }
    EAVL_FUNCTOR tuple<float> operator()(tuple<int> inputs)
    {
        ray[0] = 45.0;

        //ray.x = ray.x * factor;
        //ray.y = ray.y * factor;


        return tuple<float>(0.f);
    }

};
//-------------------------------------------------
/*
float4 ApplyTF(float* samples, int numSamples,const eavlConstTexArray<float4> *colorMap, int ncolors)
{
    float4 color= {0.0,0.0,0.0,0.0};

    for(int i=0;i < numSamples; i++)
    {
        int colorindex = float(ncolors-1) * samples[i];
        float4 c = colorMap->getValue(cmap_tref, colorindex);
        c.w *= (1.f - color.w); 
        color.x = color.x  + c.x * c.w;
        color.y = color.y  + c.y * c.w;
        color.z = color.z  + c.z * c.w;
        color.w = c.w + color.w;

        if(color.w >= 0.99)return color;
    }

    return color;
}*/
//-------------------------------------------------
struct GetPartialComposites
{   
    const eavlConstTexArray<float4> *colorMap;
    eavlView         view;
    int              nSamples;
    float*           samples;
    int              h;
    int              w;
    int              ncolors;
    float            mindepth;
    float            maxdepth;
    eavlPoint3       minComposite;
    eavlPoint3       maxComposite;
    int              zOffest;
    bool             finalPass;
    int              maxSIndx;
    int              minZPixel;

    int              dx;
    int              dy;
    //int              dz;
    int              xmin;
    int              ymin;
    eavlIntArray*             numOfPartials;
    eavlIntArray* offesetPartials;
    //int start;
    float* rays;
    //int index;
    //int origX, origY;

    GetPartialComposites( eavlView _view, int _nSamples, float* _samples,float* _rays, eavlIntArray* _offesetPartials, const eavlConstTexArray<float4> *_colorMap, int _ncolors, eavlPoint3 _minComposite, eavlPoint3 _maxComposite, int _zOffset, bool _finalPass, int _maxSIndx, int _minZPixel, int _dx, int _dy, int _xmin, int _ymin, eavlIntArray* _numOfPartials)
    : view(_view), nSamples(_nSamples), samples(_samples),rays(_rays), offesetPartials(_offesetPartials), colorMap(_colorMap), ncolors(_ncolors), minComposite(_minComposite), maxComposite(_maxComposite), finalPass(_finalPass), maxSIndx(_maxSIndx),
      dx(_dx), dy(_dy), xmin(_xmin), ymin(_ymin), numOfPartials(_numOfPartials)
    {
        w = view.w;
        h = view.h;
        zOffest = _zOffset;
        minZPixel = _minZPixel;
        

    }
 
    EAVL_FUNCTOR tuple<float> operator()(tuple<int,int> inputs )
    {
        int idx = get<0>(inputs);
        int x = idx%w;
        int y = idx/w;
        int origX = x;
        int origY = y;
        int minZsample = get<1>(inputs);
        int start = 0;
        int end =0;
        int partInd = 0;
        int index=0;
        //get the incoming color and return if the opacity is already 100%
        float4 color= {0.0,0.0,0.0,0.0};
        float4 pc = {0.0,0.0,0.0,0.0};
       // if(color.w >= 1) return tuple<float>(0.0);
        //cerr<<"Before \n";
        x-= xmin;
        y-= ymin;
        //pixel outside the AABB of the data set
        if((x >= dx) || (x < 0) || ( y >= dy) || (y < 0) || numOfPartials == 0)
        {
            return  tuple<float>(0.f);//tuple<float,float,float,float,int>(0.f,0.f,0.f,0.f, minZsample);
        }
        //cerr<<"After is \n";
        for(int z = 0 ; z < zOffest; z++)
        {
                //(x + view.w*(y + zSize*z));
            int index3d = (y*dx + x)*zOffest + z;//(x + dx*(y + dy*(z))) ;//
            int myOffest = offesetPartials->GetValue(idx);
            //if(idx == 0)
            //cerr<<" pixel "<<idx<<" myOffest is "<<myOffest<<"\n";
            //cerr<<"3D index "<<index3d<<" index "<<idx<<"\n";
            //printf("Coord = (%f,%f,%f) %d ",x,y,z, index3d);
            float value =  samples[index3d];//tsamples->getValue(samples_tref, index3d);// samples[index3d];
            
            //takes init value -1 if it was a large cell 
            //if (value <= 0.f || value > 1.f)
            //    continue; //cerr<<"Value "<<value<<"\n";
            if( value  > 0.0f)
                {  
                    if(start ==0)
                    {
                        index = myOffest*8+partInd*8;
                        
                        //rays[index+0] = idx;
                        rays[index+0] = origX;
                        rays[index+1] = origY;
                        rays[index+2] = z;
                        start = 1;
                    } //if start = 0
                    int colorindex = float(ncolors-1) * value;
                    float4 c = colorMap->getValue(cmap_tref, colorindex);
                    //cout<<"color for value "<<value<<" is "<<color.x<<" "<<color.y<<" "<<color.z<<" "<<color.w<<"\n";
                    //if(color.w < 0.95 )
                   //{c.w *= (1.f - color.w); 
                    color.x = c.x;
                    color.y = c.y;
                    color.z = c.z;
                    color.w = c.w;

                    if(pc.w< 1)
                    {
                        //c.w *= (1.f - pc.w); 
                        pc.x = pc.x  + (1-pc.w) *c.x * c.w;
                        pc.y = pc.y  + (1-pc.w) *c.y * c.w;
                        pc.z = pc.z  + (1-pc.w) *c.z * c.w;
                        pc.w = pc.w  + (1-pc.w) *c.w;

                        
                    }//pc.w < 0.95

                }// if (value >=0 )
                
            if(value < 0.0f && start == 1)
               { start = 0;
                 end = 1;
                 rays[index+3] = z-1;
                 /*
                 rays[index+4] = color.x;
                 rays[index+5] = color.y;
                 rays[index+6] = color.z;
                 rays[index+7] = color.w;*/
                 partInd++;
                //add color to arrray as partial composite
               }
               if(value >=0 && z == zOffest-1)
               {
                 start = 0;
                 end = 1;
                 rays[index+3] = z;
                 /*
                 rays[index+4] = color.x;
                 rays[index+5] = color.y;
                 rays[index+6] = color.z;
                 rays[index+7] = color.w;*/
                 partInd++;
               }

             //  if(end == 1)
               //{
                 rays[index+4] = pc.x;
                 rays[index+5] = pc.y;
                 rays[index+6] = pc.z;
                 rays[index+7] = pc.w;
                 end = 0;
                 /*
               if(rays[index+2] == rays[index+3])
               {
                 
                 //rays[index+4] = color.x;
                 //rays[index+5] = color.y;
                 //rays[index+6] = color.z;
                 //rays[index+7] = color.w;
                 rays[index+4] = pc.x;
                 rays[index+5] = pc.y;
                 rays[index+6] = pc.z;
                 rays[index+7] = pc.w;
                 end = 0;
               }
               
               else
               {
                //float4 pc = ApplyTF(partialsFloat, numOfsampperPart,colorMap,ncolors);
                 rays[index+4] = pc.x;
                 rays[index+5] = pc.y;
                 rays[index+6] = pc.z;
                 rays[index+7] = pc.w;
                 end = 0;
               }*/

             // }//if end ==1


               //if(origX == 0 && origY == 0)
                //cerr<<"Pixel 0 0 has "<<partInd<<"\n";
               /*
            int colorindex = float(ncolors-1) * value;
            float4 c = colorMap->getValue(cmap_tref, colorindex);
            //cout<<"color for value "<<value<<" is "<<color.x<<" "<<color.y<<" "<<color.z<<" "<<color.w<<"\n";
            c.w *= (1.f - color.w); 
            color.x = color.x  + c.x * c.w;
            color.y = color.y  + c.y * c.w;
            color.z = color.z  + c.z * c.w;
            color.w = c.w + color.w;

                  minZsample = min(minZsample, minZPixel + z); //we need the closest sample to get depth buffer 
            if(color.w >=1 ) break;*/

       }// for Z
        //cerr<<"Color "<<color.x<<" "<<color.y<<" "<<color.z<<" "<<color.w<<"\n";
//  cerr<<"Min Sample "<<minZsample<<"\n"; 
        //return tuple<float,float,float,float,int>(min(1.f, color.x),  min(1.f, color.y),min(1.f, color.z),min(1.f,color.w), minZsample);
     return tuple<float>(0.f);  
    }
};

//-------------------------------------------------
struct GetNumOfPartialCompNum
{   
    const eavlConstTexArray<float4> *colorMap;
    eavlView         view;
    int              nSamples;
    float*           samples;
    int              h;
    int              w;
    int              ncolors;
    float            mindepth;
    float            maxdepth;
    eavlPoint3       minComposite;
    eavlPoint3       maxComposite;
    int              zOffest;
    bool             finalPass;
    int              maxSIndx;
    int                    minZPixel;

    int              dx;
    int              dy;
    //int              dz;
    int              xmin;
    int              ymin;
    //int              start;
    GetNumOfPartialCompNum( eavlView _view, int _nSamples, float* _samples, const eavlConstTexArray<float4> *_colorMap, int _ncolors, eavlPoint3 _minComposite, eavlPoint3 _maxComposite, int _zOffset, bool _finalPass, int _maxSIndx, int _minZPixel, int _dx, int _dy, int _xmin, int _ymin)
    : view(_view), nSamples(_nSamples), samples(_samples), colorMap(_colorMap), ncolors(_ncolors), minComposite(_minComposite), maxComposite(_maxComposite), finalPass(_finalPass), maxSIndx(_maxSIndx),
      dx(_dx), dy(_dy), xmin(_xmin), ymin(_ymin)
    {
        w = view.w;
        h = view.h;
        zOffest = _zOffset;
        minZPixel = _minZPixel;
        
    }
 
    EAVL_FUNCTOR tuple<int> operator()(tuple< int> inputs )
    {
        int idx = get<0>(inputs);
        int x = idx%w;
        int y = idx/w;
        int numOfPartials = 0;
        int start = 0;
        //get the incoming color and return if the opacity is already 100%
        //cerr<<"Before \n";
        x-= xmin;
        y-= ymin;
        //pixel outside the AABB of the data set
        if((x >= dx) || (x < 0) || ( y >= dy) || (y < 0))
        {
            return tuple<int>(0);
        }
        //cerr<<"After is \n";
        for(int z = 0 ; z < zOffest; z++)
        {
                //(x + view.w*(y + zSize*z));
            int index3d = (y*dx + x)*zOffest + z;//(x + dx*(y + dy*(z))) ;//
            
            //printf("Coord = (%f,%f,%f) %d ",x,y,z, index3d);
            float value =  samples[index3d];//tsamples->getValue(samples_tref, index3d);// samples[index3d];
            
            //takes init value -1 if it was a large cell 
           // if (value <= 0.f || value > 1.f)
              //  continue; //cerr<<"Value "<<value<<"\n";
          //  if(value < 0)
            //cerr<<"Value "<<value<<" at z "<<z<<"\n";


            if( value  >= 0.0f  && start == 0)
                {  
                    start = 1;
                    numOfPartials++;
                    
                }
            if(value < 0.0f && start == 1)
                start = 0;
        
            /*
            int colorindex = float(ncolors-1) * value;
            float4 c = colorMap->getValue(cmap_tref, colorindex);
            //cout<<"color for value "<<value<<" is "<<color.x<<" "<<color.y<<" "<<color.z<<" "<<color.w<<"\n";
            c.w *= (1.f - color.w); 
            color.x = color.x  + c.x * c.w;
            color.y = color.y  + c.y * c.w;
            color.z = color.z  + c.z * c.w;
            color.w = c.w + color.w;

                  minZsample = min(minZsample, minZPixel + z); //we need the closest sample to get depth buffer 
            if(color.w >=1 ) break;*/
       }
    
       //cerr<<"Num of partials in this pixel is "<<numOfPartials<<"\n";
//  cerr<<"Min Sample "<<minZsample<<"\n"; 
        return tuple<int>(numOfPartials);
        
    }
};

//-------------------------------------------------

//compisite the bakground color into the framebuffer
struct CompositeBG
{   
    float4 cc;
    CompositeBG(eavlColor &_bgColor)
    {
        cc.x = _bgColor.c[0];
        cc.y = _bgColor.c[1];
        cc.z = _bgColor.c[2];
        cc.w = _bgColor.c[3]; 
        
        
    }

    EAVL_FUNCTOR tuple<float,float,float,float> operator()(tuple<float, float, float, float> inputs )
    {

        float4 color= {get<0>(inputs),get<1>(inputs),get<2>(inputs),get<3>(inputs)};
        if(color.w >= 1) return tuple<float,float,float,float>(color.x, color.y, color.z,color.w);

        float4 c = cc; 
        
        c.w *= (1.f - color.w); 
        color.x = color.x  + c.x * c.w;
        color.y = color.y  + c.y * c.w;
        color.z = color.z  + c.z * c.w;
        color.w = c.w + color.w;

        return tuple<float,float,float,float>(min(1.f, color.x),  min(1.f, color.y),min(1.f, color.z),min(1.f,color.w) );
    }
};

//-------------------------------------------------

eavlFloatArray* eavlSimpleVRMutator::getDepthBuffer(float proj22, float proj23, float proj32)
{ 

        eavlExecutor::AddOperation(new_eavlMapOp(eavlOpArgs(minSample), eavlOpArgs(zBuffer), convertDepthFunctor(view,nSamples)),"convertDepth");
        eavlExecutor::Go();
        return zBuffer;
}

//-------------------------------------------------

void eavlSimpleVRMutator::setColorMap3f(float* cmap,int size)
{
    if(verbose) cout<<"Setting new color map 3f"<<endl;
    colormapSize = size;
    if(color_map_array != NULL)
    {
        color_map_array->unbind(cmap_tref);
        
        delete color_map_array;
    
        color_map_array = NULL;
    }
    if(colormap_raw != NULL)
    {
        delete[] colormap_raw;
        colormap_raw = NULL;
    }
    colormap_raw= new float[size*4];
    
    for(int i=0;i<size;i++)
    {
        colormap_raw[i*4  ] = cmap[i*3  ];
        colormap_raw[i*4+1] = cmap[i*3+1];
        colormap_raw[i*4+2] = cmap[i*3+2];
        colormap_raw[i*4+3] = .01f;          //test Alpha
    }
    color_map_array = new eavlConstTexArray<float4>((float4*)colormap_raw, colormapSize, cmap_tref, cpu);
}

//-------------------------------------------------

void eavlSimpleVRMutator::setColorMap4f(float* cmap,int size)
{
    if(verbose) cout<<"Setting new color map of size "<<size<<endl;
    colormapSize = size;
    if(color_map_array != NULL)
    {
        color_map_array->unbind(cmap_tref);
        
        delete color_map_array;
    
        color_map_array = NULL;
    }
    if(colormap_raw != NULL)
    {
        delete[] colormap_raw;
        colormap_raw = NULL;
    }
    colormap_raw= new float[size*4];
    
    for(int i=0;i<size;i++)
    {
        colormap_raw[i*4  ] = cmap[i*4  ];
        colormap_raw[i*4+1] = cmap[i*4+1];
        colormap_raw[i*4+2] = cmap[i*4+2];
        colormap_raw[i*4+3] = cmap[i*4+3];  
        //cout<<"Color "<<colormap_raw[i*4  ]<<" "<<colormap_raw[i*4 +1]<<" "<<colormap_raw[i*4 +2]<<" "<<colormap_raw[i*4 +3]<<endl;        
    }
    color_map_array = new eavlConstTexArray<float4>((float4*)colormap_raw, colormapSize, cmap_tref, cpu);
}

//-------------------------------------------------

void eavlSimpleVRMutator::setDefaultColorMap()
{   if(verbose) cout<<"setting defaul color map"<<endl;
    if(color_map_array!=NULL)
    {
        color_map_array->unbind(cmap_tref);
        delete color_map_array;
        color_map_array = NULL;
    }
    if(colormap_raw!=NULL)
    {
        delete[] colormap_raw;
        colormap_raw = NULL;
    }
    //two values all 1s
    colormapSize=2;
    colormap_raw= new float[8];
    for(int i=0;i<8;i++) colormap_raw[i]=1.f;
    color_map_array = new eavlConstTexArray<float4>((float4*)colormap_raw, colormapSize, cmap_tref, cpu);
    if(verbose) cout<<"Done setting defaul color map"<<endl;

}

//-------------------------------------------------

void eavlSimpleVRMutator::calcMemoryRequirements()
{

    unsigned long int mem = 0; //mem in bytes

    mem += pixelsPerPass * sizeof(float);       //samples
    mem += numTets * 12 * sizeof(float);
    mem += height * width * 4 * sizeof(float);  //framebuffer
    mem += height * width * sizeof(float);      //zbuffer
    mem += numTets * 4 * sizeof(float);         //scalars
    mem += numTets * 2;                         //min and max passes (BYTEs)
    mem += numTets * sizeof(int);               //interator
    mem += height * width * sizeof(int);        //screen iterator;
    mem += passCountEstimate * 12 * sizeof(float);//screen space coords
    //find pass members arrays
    mem += numTets * 4 * sizeof(int);           //indexscan, mask, currentPassMembers
    mem += passCountEstimate * sizeof(int);     //reverse index
    double memd = (double) mem / (1024.0 * 1024.0);
    if(verbose) printf("Memory needed %10.3f MB. Do you have enough?\n", memd);
    
    if(!cpu)
    {

#ifdef HAVE_CUDA
        size_t free_byte;
        size_t total_byte;
        hipMemGetInfo( &free_byte, &total_byte );
        double free_db = (double)free_byte ;
        double total_db = (double)total_byte ;
        double used_db = total_db - free_db ;
        if(verbose) printf("GPU memory usage: used = %f, free = %f MB, total = %f MB\n", used_db/1024.0/1024.0, free_db/1024.0/1024.0, total_db/1024.0/1024.0);
        if(mem > free_byte)
        {
            cout<<"Warning : this will exceed memory usage by "<< (mem - free_byte) << "bytes.\n";
        }
#endif

    }   
}

//-------------------------------------------------

void printGPUMemUsage()
{
    #ifdef HAVE_CUDA
        size_t free_byte;
        size_t total_byte;
        hipMemGetInfo( &free_byte, &total_byte );
        double free_db = (double)free_byte ;
        double total_db = (double)total_byte ;
        double used_db = total_db - free_db ;
        printf("GPU memory usage: used = %f, free = %f MB, total = %f MB\n", used_db/1024.0/1024.0, free_db/1024.0/1024.0, total_db/1024.0/1024.0);
#endif
}

//-------------------------------------------------

void eavlSimpleVRMutator::clearSamplesArray()
{
    //cerr<<"In function clearSamplesArray\n";
    int clearValue = 0xbf800000; //-1 float
    size_t bytes = pixelsPerPass * sizeof(float);
    if(!cpu)
    {
#ifdef HAVE_CUDA
       hipMemset(samples->GetCUDAArray(), clearValue,bytes);
       CUDA_CHECK_ERROR();
#endif
    }
    else
    {
       memset(samples->GetHostArray(), clearValue, bytes);   
    }


}

//-------------------------------------------------

void eavlSimpleVRMutator::init()
{
    
    if(sizeDirty)
    {   
        setNumPasses(numPasses);
        if(verbose) cout<<"Size Dirty"<<endl;
       
        deleteClassPtr(samples);
        deleteClassPtr(framebuffer);
        deleteClassPtr(zBuffer);
        deleteClassPtr(minSample);
        
        samples         = new eavlFloatArray("",1,pixelsPerPass);
        framebuffer     = new eavlFloatArray("",1,height*width*4);
        rgba            = new eavlByteArray("",1,height*width*4);
        zBuffer         = new eavlFloatArray("",1,height*width);
        minSample       = new eavlIntArray("",1,height*width);
        clearSamplesArray();
        if(verbose) cout<<"Samples array size "<<pixelsPerPass<<" Current CPU val "<<cpu<< endl;
        if(verbose) cout<<"Current framebuffer size "<<(height*width*4)<<endl;
        sizeDirty = false;
         
    }

    if(geomDirty && numTets > 0)
    {   
        if(verbose) cout<<"Geometry Dirty"<<endl;
        firstPass = true;
        passNumDirty = true;
        freeTextures();
        freeRaw();

        deleteClassPtr(minPasses);
        deleteClassPtr(maxPasses);
        deleteClassPtr(iterator);
        deleteClassPtr(dummy);
        deleteClassPtr(indexScan);
        deleteClassPtr(mask);

        tetSOA = scene->getEavlTetPtrs();
        
        scalars_array       = new eavlConstTexArray<float4>( (float4*) scene->getScalarPtr()->GetHostArray(), 
                                                             numTets, 
                                                             scalars_tref, 
                                                             cpu);
        minPasses = new eavlByteArray("",1, numTets);
        maxPasses = new eavlByteArray("",1, numTets);
        indexScan = new eavlIntArray("",1, numTets);
        mask = new eavlIntArray("",1, numTets);
        sumSamples = new eavlIntArray("",1,numTets);
        iterator      = new eavlIntArray("",1, numTets);
        dummy = new eavlFloatArray("",1,1); //wtf
        for(int i=0; i < numTets; i++) iterator->SetValue(i,i);
        //readTransferFunction(tfFilename);
        geomDirty = false;
    }

    //we are trying to keep the mem usage down. We will conservativily estimate the number of
    //indexes to keep in here. Edge case would we super zoomed in a particlar region which
    //would maximize the wasted space.
    
    if(!firstPass)
    {
        float ratio = maxPassSize / (float) passCountEstimate;
        if(ratio < .9 || ratio > 1.f) 
        {
            passCountEstimate = maxPassSize + (int)(maxPassSize * .1); //add a little padding here.
            passNumDirty = true;
            cout<<"Ajdusting Pass size"<<endl;
        }
    }

    if(passNumDirty)
    {
        if(verbose) cout<<"Pass Dirty"<<endl;
        if(firstPass) 
        {
       
            passCountEstimate = (int)((numTets / numPasses) * PASS_ESTIMATE_FACTOR); //TODO: see how close we can cut this
            if(numPasses == 1) passCountEstimate = numTets;
            maxPassSize =-1;
            firstPass = false;
        }
        deleteClassPtr(currentPassMembers);
        deleteClassPtr(reverseIndex);
        deleteClassPtr(ssa);
        deleteClassPtr(ssb);
        deleteClassPtr(ssc);
        deleteClassPtr(ssd);
        deleteClassPtr(screenIterator);
        if(false && numPasses == 1)
        {
            currentPassMembers = iterator;
        }
        else
        {   //we don't need to allocate this if we are only doing one pass
            currentPassMembers = new eavlIntArray("",1, passCountEstimate);
            reverseIndex = new eavlIntArray("",1, passCountEstimate); 
        }
        int size = width * height;
        screenIterator  = new eavlIntArray("",1,size);
        for(int i=0; i < size; i++) screenIterator->SetValue(i,i);
        int space  = passCountEstimate*3;
        if(space < 0) cout<<"ERROR int overflow"<<endl;
        if(verbose) cout<<"allocating pce "<<passCountEstimate<<endl;
        ssa = new eavlFloatArray("",1, passCountEstimate*3); 
        ssb = new eavlFloatArray("",1, passCountEstimate*3);
        ssc = new eavlFloatArray("",1, passCountEstimate*3);
        ssd = new eavlFloatArray("",1, passCountEstimate*3);
        passNumDirty = false;
    }
    
    calcMemoryRequirements();
}

//-------------------------------------------------

struct PassThreshFunctor
{
    int passId;
    PassThreshFunctor(int _passId) : passId(_passId)
    {}

    EAVL_FUNCTOR tuple<int> operator()(tuple<int,int> input){
        int minp = get<0>(input);
        int maxp = get<1>(input);
        if((minp <= passId) && (maxp >= passId)) return tuple<int>(1);
        else return tuple<int>(0);
    }
};

//-------------------------------------------------

void eavlSimpleVRMutator::performScreenSpaceTransform(eavlIntArray *tetIds, int number)
{
    //cerr<<"IN PerformScreen\n";
	int numPassMembers = tetIds->GetNumberOfTuples();
    int outputArraySize = ssa->GetNumberOfTuples() / 3;
   
  // cerr<<"Number of Big Cells "<<numPassMembers<<"\n"; 

   if(numPassMembers > outputArraySize)
    {
        cout<<"WARNING!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n";
        cout<<"Too many input cells for screen space transform\n";
        exit(1);
    }

    eavlExecutor::AddOperation(new_eavlMapOp(eavlOpArgs(tetIds),
                                                     eavlOpArgs(eavlIndexable<eavlFloatArray>(ssa,*i1),
                                                                eavlIndexable<eavlFloatArray>(ssa,*i2),
                                                                eavlIndexable<eavlFloatArray>(ssa,*i3),
                                                                eavlIndexable<eavlFloatArray>(ssb,*i1),
                                                                eavlIndexable<eavlFloatArray>(ssb,*i2),
                                                                eavlIndexable<eavlFloatArray>(ssb,*i3),
                                                                eavlIndexable<eavlFloatArray>(ssc,*i1),
                                                                eavlIndexable<eavlFloatArray>(ssc,*i2),
                                                                eavlIndexable<eavlFloatArray>(ssc,*i3),
                                                                eavlIndexable<eavlFloatArray>(ssd,*i1),
                                                                eavlIndexable<eavlFloatArray>(ssd,*i2),
                                                                eavlIndexable<eavlFloatArray>(ssd,*i3)),
                                                    ScreenSpaceFunctor(xtet,ytet,ztet,view, nSamples, xmin,ymin,zmin),number),
                                                    "Screen Space transform");
    
	
    //cerr<<"AddOperation done\n";
	eavlExecutor::Go();
	//cerr<<"Executor done\n";
}

void eavlSimpleVRMutator::findCurrentPassMembers(int pass)
{
    int passtime;
    if(verbose)  passtime = eavlTimer::Start();

    eavlExecutor::AddOperation(new_eavlMapOp(eavlOpArgs(minPasses,maxPasses),
                                         eavlOpArgs(mask),
                                         PassThreshFunctor(pass)),
                                         "find pass members");

    eavlExecutor::Go();
    eavlExecutor::AddOperation(new eavlPrefixSumOp_1(mask,indexScan,false), //inclusive==true exclusive ==false
                                                     "create indexes");
    eavlExecutor::Go();

    eavlExecutor::AddOperation(new eavlReduceOp_1<eavlAddFunctor<int> >
                              (mask,
                               passCount,
                               eavlAddFunctor<int>()),
                               "count output");
    eavlExecutor::Go();

    passSize = passCount->GetValue(0);
    maxPassSize = max(maxPassSize, passSize);

    if(passSize > passCountEstimate)
    {
      cout<<"WARNING Exceeded max passSize:  maxPassSize "<<maxPassSize<<" estimate "<<passCountEstimate<<endl;  
      passNumDirty = true;
      THROW(eavlException, "exceeded max pass size.");
    } 

    if(passSize == 0)
    {
        return;
    }
    
    eavlExecutor::AddOperation(new eavlSimpleReverseIndexOp(mask,
                                                            indexScan,
                                                            reverseIndex),
                                                            "generate reverse lookup");
    eavlExecutor::Go();
    
    eavlExecutor::AddOperation(new_eavlGatherOp(eavlOpArgs(iterator),
                                                eavlOpArgs(currentPassMembers),
                                                eavlOpArgs(reverseIndex),
                                                passSize),
                                                "pull in the tets for this pass");
    eavlExecutor::Go();
    

    if(verbose) passSelectionTime += eavlTimer::Stop(passtime,"pass");
}

//-------------------------------------------------
void  eavlSimpleVRMutator::Execute()
{
    //
    // If we are doing parallel compositing, we just want the partial
    // composites without the background color
    //
    //cout<<view.P<<" \n"<<view.V<<endl;
    // view.SetupMatrices();
    // cout<<view.P<<" \n"<<view.V<<endl;
    //cerr<<"IN execute\n";
    if(isTransparentBG) 
    {
        bgColor.c[0] =0.f; 
        bgColor.c[1] =0.f; 
        bgColor.c[2] =0.f; 
        bgColor.c[3] =0.f;
    }

    //timing accumulators
    double clearTime = 0;
    passFilterTime = 0;
    compositeTime = 0;
    passSelectionTime = 0;
    sampleTime = 0;
    allocateTime = 0;
    screenSpaceTime = 0;
    renderTime = 0;
   
    int tets = scene->getNumTets();
    //eavlPartialComp* rays; 
   
    if(tets != numTets)
    {
        geomDirty = true;
        numTets = tets;
    }
    if(verbose) 
       cout<<"Num Tets = "<<numTets<<endl;

    // Pixels extents are used to skip empty space in compositing
    // and for allocating sample buffer
    eavlPoint3 mins(scene->getSceneBBox().min.x,scene->getSceneBBox().min.y,scene->getSceneBBox().min.z);
    eavlPoint3 maxs(scene->getSceneBBox().max.x,scene->getSceneBBox().max.y,scene->getSceneBBox().max.z);
    getBBoxPixelExtent(mins,maxs);
    //
    //  Set sample buffer information
    //
    xmin = mins.x;
    ymin = mins.y;
    zmin = mins.z;
    int new_dx = maxs.x - mins.x;
    int new_dy = maxs.y - mins.y;
    int new_dz = maxs.z - mins.z;
    //cerr<<"Before if sizeDirty\n";
    if(new_dx != dx || new_dy != dy || new_dz != dz) sizeDirty = true;
    dx = new_dx;
    dy = new_dy;
    dz = new_dz;
    //cerr<<"After sizeDirty\n";
    int tinit;
    if(verbose) tinit = eavlTimer::Start();
    init();
    //cerr<<"After init\n";
    //cerr<<"num of tets "<<tets<<"\n";
    if(tets < 1)
    {
        //There is nothing to render. Set depth and framebuffer
        eavlExecutor::AddOperation(new_eavlMapOp(eavlOpArgs(minSample),
                                             eavlOpArgs(minSample),
                                             IntMemsetFunctor(nSamples+1000)), //what should this be?
                                             "clear first sample");
        eavlExecutor::Go();

        //cerr<<"clear first sample for tets <0 \n";
        
        eavlExecutor::AddOperation(new_eavlMapOp(eavlOpArgs(framebuffer),
                                             eavlOpArgs(framebuffer),
                                             FloatMemsetFunctor(0)),
                                             "clear Frame Buffer");
        eavlExecutor::Go();

        //cerr<<"clear Frame Buffer for tets <0 \n";

        eavlExecutor::AddOperation(new_eavlMapOp(eavlOpArgs(eavlIndexable<eavlFloatArray>(framebuffer,*ir),
                                                             eavlIndexable<eavlFloatArray>(framebuffer,*ig),
                                                             eavlIndexable<eavlFloatArray>(framebuffer,*ib),
                                                             eavlIndexable<eavlFloatArray>(framebuffer,*ia)),
                                                  eavlOpArgs(eavlIndexable<eavlFloatArray>(framebuffer,*ir),
                                                             eavlIndexable<eavlFloatArray>(framebuffer,*ig),
                                                             eavlIndexable<eavlFloatArray>(framebuffer,*ib),
                                                             eavlIndexable<eavlFloatArray>(framebuffer,*ia)),
                                                 CompositeBG(bgColor), height*width),
                                                 "Composite");
        eavlExecutor::Go();

        //cerr<<"Composite for tets <0 \n";
        return;
    }
    
    
    //cerr<<"Before cpu gpu stuff \n";
    if(!cpu)
    {
        //cout<<"Getting cuda array for tets."<<endl;
        xtet = (float4*) tetSOA[0]->GetCUDAArray();
        ytet = (float4*) tetSOA[1]->GetCUDAArray();
        ztet = (float4*) tetSOA[2]->GetCUDAArray();
    }
    else 
    {
        xtet = (float4*) tetSOA[0]->GetHostArray();
        ytet = (float4*) tetSOA[1]->GetHostArray();
        ztet = (float4*) tetSOA[2]->GetHostArray();
    }
    float* samplePtr;
    //PartialComposite* raysPtr;
    if(!cpu)
    {
        samplePtr = (float*) samples->GetCUDAArray();
        //raysPtr      = (PartialComposite*) rays->GetCUDAArray();
    }
    else 
    {
        samplePtr = (float*) samples->GetHostArray();
        //raysPtr      = (PartialComposite*) rays->GetCUDAArray();
    }

    float* alphaPtr;
    if(!cpu)
    {
        alphaPtr = (float*) framebuffer->GetCUDAArray();
    }
    else 
    {
        alphaPtr = (float*) framebuffer->GetHostArray();
    }
    if(verbose) cout<<"Init        RUNTIME: "<<eavlTimer::Stop(tinit,"init")<<endl;

    int ttot;
    if(verbose) ttot = eavlTimer::Start();

    if(verbose)
    {
        cout<<"BBox Screen Space "<<mins<<maxs<<endl; 
    }
    int tclear;
    if(verbose) tclear = eavlTimer::Start();

    //cerr<<"Bfore adding operations \n";
    eavlExecutor::AddOperation(new_eavlMapOp(eavlOpArgs(framebuffer),
                                             eavlOpArgs(framebuffer),
                                             FloatMemsetFunctor(0)),
                                             "clear Frame Buffer");
    eavlExecutor::Go();

    eavlExecutor::AddOperation(new_eavlMapOp(eavlOpArgs(zBuffer),
                                             eavlOpArgs(zBuffer),
                                             FloatMemsetFunctor(1.f)),
                                             "clear Frame Buffer");
    eavlExecutor::Go();
    
     eavlExecutor::AddOperation(new_eavlMapOp(eavlOpArgs(minSample),
                                             eavlOpArgs(minSample),
                                             IntMemsetFunctor(nSamples+1000)), //TODO:Maybe this should be higher
                                             "clear first sample");
    eavlExecutor::Go();
   
    
    
     if(verbose) 
        cout<<"ClearBuffs  RUNTIME: "<<eavlTimer::Stop(tclear,"")<<endl;

    int ttrans;
    if(verbose) ttrans = eavlTimer::Start();
    if(false && numPasses == 1)
    {
        //just set all tets to the first pass
        eavlExecutor::AddOperation(new_eavlMapOp(eavlOpArgs(minPasses),
                                             eavlOpArgs(minPasses),
                                             IntMemsetFunctor(0)),
                                             "set");
        eavlExecutor::Go();
        eavlExecutor::AddOperation(new_eavlMapOp(eavlOpArgs(maxPasses),
                                             eavlOpArgs(maxPasses),
                                             IntMemsetFunctor(0)),
                                             "set");
        eavlExecutor::Go();
        //passSize = numTets;
    }
    else
    {
        //find the min and max passes the tets belong to
        cerr<<"Calling PassRange with sampleLCFlag value = "<<sampleLCFlag<<"\n";
        eavlExecutor::AddOperation(new_eavlMapOp(eavlOpArgs(iterator),
                                             eavlOpArgs(minPasses, maxPasses,sumSamples),
                                             PassRange(xtet,ytet,ztet, view, nSamples, numPasses, zmin,dz,sampleLCFlag)),
                                             "PassFilter");
        eavlExecutor::Go(); 
    }
    

    if(verbose) passFilterTime =  eavlTimer::Stop(ttrans,"ttrans");
        
    
    //cout<<"Pass Z stride "<<passZStride<<endl;
    for(int i = 0; i < numPasses; i++)
    {
        // ins sample space
        int pixelZMin = passZStride * i;
        int pixelZMax = passZStride * (i + 1) - 1;
      
        try
        {
            //if(numPasses > 1) 
                findCurrentPassMembers(i);
        }
        catch(eavlException &e)
        {
            return;
        }
        
        //cerr<<"Pass size "<<passSize<<"\n";
        
        if(passSize > 0)
        {

            int tclearS;
            if(verbose) tclearS = eavlTimer::Start();
            if (i != 0) clearSamplesArray();  //this is a win on CPU for sure, gpu seems to be the same
            //cerr<<"clearSamplesArray is done\n";
            // eavlExecutor::AddOperation(new_eavlMapOp(eavlOpArgs(samples),
            //                                          eavlOpArgs(samples),
            //                                          FloatMemsetFunctor(-1.f)),
            //                                          "clear Frame Buffer");
            // eavlExecutor::Go();
            if(verbose) clearTime += eavlTimer::Stop(tclearS,"");
                
            int tsspace;
            if(verbose) tsspace = eavlTimer::Start();
            
           // cerr<<"Before screen space transformation\n";
            performScreenSpaceTransform(currentPassMembers,passSize);

	        //cerr<<"Done Screen Space Transform\n";
    
            if(verbose) screenSpaceTime += eavlTimer::Stop(tsspace,"sample");
            int tsample;
            if(verbose) tsample = eavlTimer::Start();
            //Call Sample function
            eavlExecutor::AddOperation(new_eavlMapOp(eavlOpArgs(eavlIndexable<eavlIntArray>(currentPassMembers),
                                                        eavlIndexable<eavlFloatArray>(ssa,*i1),
                                                        eavlIndexable<eavlFloatArray>(ssa,*i2),
                                                        eavlIndexable<eavlFloatArray>(ssa,*i3),
                                                        eavlIndexable<eavlFloatArray>(ssb,*i1),
                                                        eavlIndexable<eavlFloatArray>(ssb,*i2),
                                                        eavlIndexable<eavlFloatArray>(ssb,*i3),
                                                        eavlIndexable<eavlFloatArray>(ssc,*i1),
                                                        eavlIndexable<eavlFloatArray>(ssc,*i2),
                                                        eavlIndexable<eavlFloatArray>(ssc,*i3),
                                                        eavlIndexable<eavlFloatArray>(ssd,*i1),
                                                        eavlIndexable<eavlFloatArray>(ssd,*i2),
                                                        eavlIndexable<eavlFloatArray>(ssd,*i3)),
                                                        eavlOpArgs(eavlIndexable<eavlFloatArray>(dummy,*idummy)), 
                                                     SampleFunctor3(scalars_array, view, nSamples, samplePtr, pixelZMin, pixelZMax, passZStride, alphaPtr, dx, dy,dz, xmin,ymin),passSize),
                                                     "Sampler");
           eavlExecutor::Go();
            //cerr<<"  Done Sampling \n";


            if(verbose) sampleTime += eavlTimer::Stop(tsample,"sample");
            int talloc;
            if(verbose) talloc = eavlTimer::Start();

            if(verbose) allocateTime += eavlTimer::Stop(talloc,"sample");
            //eavlArrayIndexer * ifb = new eavlArrayIndexer(1, offset);
            //cout<<"screenIterator last value "<<screenIterator->GetS
            bool finalPass = (i == numPasses - 1) ? true : false;
            int tcomp;
            if(verbose) tcomp = eavlTimer::Start();
      
            numOfPartials = new eavlIntArray("",1,width*height);
            //cerr<<"**** pixel 0,0 "<<numOfPartials->GetValue(0)<<"\n";
            eavlExecutor::AddOperation(new_eavlMapOp(eavlOpArgs(screenIterator),
                                             eavlOpArgs(numOfPartials),
                                             GetNumOfPartialCompNum( view, nSamples, samplePtr, color_map_array, colormapSize, mins, maxs, passZStride, finalPass, pixelsPerPass,pixelZMin, dx,dy,xmin,ymin), width*height),
                                             "number of partials");


            eavlExecutor::Go();

            //cerr<<"Num of components "<<numOfPartials->GetNumberOfTuples()<<"\n";
            //for(int i=0; i< numOfPartials->GetNumberOfTuples(); i++)
               // if(numOfPartials->GetValue(i)!=0)
                //cerr<<"Num of partials for item "<<i<<" is "<<numOfPartials->GetValue(i)<<"\n";


             totalNumberOfPArtials = new eavlIntArray("",1,1); 
             //totalNumberOfPArtials = 0;
             eavlExecutor::AddOperation(new eavlReduceOp_1<eavlAddFunctor<int> >
                              (numOfPartials,
                               totalNumberOfPArtials,
                               eavlAddFunctor<int>()),
                               "count total number of partials");

             eavlExecutor::Go();
             //cerr<<"Actual data size "<<dx*dy<<"\n";
             //cerr<<"Total number of partials "<<totalNumberOfPArtials->GetValue(0)<<"\n";

            offesetPartials = new eavlIntArray("",1,width*height);

            //False = exclusive scan output counts to get output index
            //IMPORTANT: I want to set it to false because true generates errors
            //Explination on my Doc
            eavlExecutor::AddOperation(new eavlPrefixSumOp_1(numOfPartials,
                              offesetPartials,
                              false),
                            "scan to generate starting out offeset");
                            
            eavlExecutor::Go();

            
            int raySize = totalNumberOfPArtials->GetValue(0) * 8;
            //cerr<<"Ray size "<<raySize<<"\n";
            myFloatrays = new eavlFloatArray("",1, raySize);
            
            float* raysPtr;
             if(!cpu) raysPtr      = (float*) myFloatrays->GetCUDAArray();
            else      raysPtr      = (float*) myFloatrays->GetHostArray();

            
            /*
            eavlExecutor::AddOperation(new_eavlMapOp(eavlOpArgs(eavlIndexable<eavlIntArray>(currentPassMembers)),
                                                     eavlOpArgs(eavlIndexable<eavlFloatArray>(dummy,*idummy)),
                                                     TestMyStruct(raysPtr,offesetPartials)),
                                                     "Test Map on rays");

            eavlExecutor::Go();*/

            //cerr<<"******** Test my Functor "<<myFloatrays->GetValue(0)<<"\n";

    
            eavlExecutor::AddOperation(new_eavlMapOp(eavlOpArgs(eavlIndexable<eavlIntArray>(screenIterator),
                                                                eavlIndexable<eavlIntArray>(minSample)),
                                                       eavlOpArgs(eavlIndexable<eavlFloatArray>(dummy,*idummy)),
                                                       GetPartialComposites( view, nSamples, samplePtr,raysPtr,offesetPartials,  color_map_array, colormapSize, mins, maxs, passZStride, finalPass, pixelsPerPass,pixelZMin, dx,dy,xmin,ymin,numOfPartials), width*height),
                                                       "Get Partial Composite");

            eavlExecutor::Go();

           //cerr<<"Got Partials \n";
            
             
            //-----------------------------------------------
            
            
             eavlExecutor::AddOperation(new_eavlMapOp(eavlOpArgs(eavlIndexable<eavlIntArray>(screenIterator),
                                                                 eavlIndexable<eavlFloatArray>(framebuffer,*ir),
                                                                 eavlIndexable<eavlFloatArray>(framebuffer,*ig),
                                                                 eavlIndexable<eavlFloatArray>(framebuffer,*ib),
                                                                 eavlIndexable<eavlFloatArray>(framebuffer,*ia),
                                                                 eavlIndexable<eavlIntArray>(minSample)),
                                                      eavlOpArgs(eavlIndexable<eavlFloatArray>(framebuffer,*ir),
                                                                 eavlIndexable<eavlFloatArray>(framebuffer,*ig),
                                                                 eavlIndexable<eavlFloatArray>(framebuffer,*ib),
                                                                 eavlIndexable<eavlFloatArray>(framebuffer,*ia),
                                                                 eavlIndexable<eavlIntArray>(minSample)),
                                                     CompositeFunctorFB( view, nSamples, samplePtr, color_map_array, colormapSize, mins, maxs, passZStride, finalPass, pixelsPerPass,pixelZMin, dx,dy,xmin,ymin), width*height),
                                                     "Composite");


            
	    //cerr<<"Add composite op\n";
	    eavlExecutor::Go();
	    //cerr<<"Done composite \n";
            if(verbose) compositeTime += eavlTimer::Stop(tcomp,"tcomp");

	   // cerr<<"Done composite \n";

        }//if(passSize > 0)
        else 
            {   //Did this to avoid having Segmentation fault when passSize = 0
                //Check with Matt, use the testvolume example with all LC
                myFloatrays = new eavlFloatArray("",1, 0);
                //To avoid having Segmentation fault when passSize = 0
                totalNumberOfPArtials = new eavlIntArray("",1,1);
            }
    }//for each pass
    if(verbose) renderTime  = eavlTimer::Stop(ttot,"total render");
    if(verbose) cout<<"PassFilter  RUNTIME: "<<passFilterTime<<endl;
   // cout<<"Clear Sample  RUNTIME: "<<clearTime<<endl;
    if(verbose) cout<<"PassSel     RUNTIME: "<<passSelectionTime<<" Pass AVE: "<<passSelectionTime / (float)numPasses<<endl;
    if(verbose) cout<<"ScreenSpace RUNTIME: "<<screenSpaceTime<<" Pass AVE: "<<screenSpaceTime / (float)numPasses<<endl;
    if(verbose) cout<<"Sample      RUNTIME: "<<sampleTime<<" Pass AVE: "<<sampleTime / (float)numPasses<<endl;
    if(verbose) cout<<"Composite   RUNTIME: "<<compositeTime<<" Pass AVE: "<<compositeTime / (float)numPasses<<endl;
    if(verbose) cout<<"Alloc       RUNTIME: "<<allocateTime<<" Pass AVE: "<<allocateTime / (float)numPasses<<endl;
    if(verbose) cout<<"Total       RUNTIME: "<<renderTime<<endl;
    //dataWriter();
    //composite my pixel color with background

    //cerr<<"Before composite\n";
 eavlExecutor::AddOperation(new_eavlMapOp(eavlOpArgs(
                                                                 eavlIndexable<eavlFloatArray>(framebuffer,*ir),
                                                                 eavlIndexable<eavlFloatArray>(framebuffer,*ig),
                                                                 eavlIndexable<eavlFloatArray>(framebuffer,*ib),
                                                                 eavlIndexable<eavlFloatArray>(framebuffer,*ia)),
                                                      eavlOpArgs(eavlIndexable<eavlFloatArray>(framebuffer,*ir),
                                                                 eavlIndexable<eavlFloatArray>(framebuffer,*ig),
                                                                 eavlIndexable<eavlFloatArray>(framebuffer,*ib),
                                                                 eavlIndexable<eavlFloatArray>(framebuffer,*ia)),
                                                     CompositeBG(bgColor), height*width),
                                                     "Composite");
    eavlExecutor::Go();

    //cerr<<"After composte \n";
}


inline bool exists (const std::string& name) {
    ifstream f(name.c_str());
    if (f.good()) {
        f.close();
        return true;
    } else {
        f.close();
        return false;
    }   
}

//-------------------------------------------------

void  eavlSimpleVRMutator::dataWriter()
{
  string sCPU = "_CPU_";
  string sGPU = "_GPU_";
  string dfile;
  if(cpu) dfile = "datafile_" + sCPU + dataname + ".dat";
  else dfile = "datafile_" + sGPU + dataname + ".dat";  
   
  if(!exists(dfile))
  {
    ofstream boilerplate;
    boilerplate.open (dfile.c_str());
    boilerplate << "Step\n";
    boilerplate << "Pass Filter\n";
    boilerplate << "Pass Selection\n";
    boilerplate << "Screen Space\n";
    boilerplate << "Sampling\n";
    boilerplate << "Compostiting\n";
    boilerplate << "Render\n";
    boilerplate.close();
  }
  string separator = ",";
  string line[7];
  double times[6];
  times[0] = passFilterTime;
  times[1] = passSelectionTime;
  times[2] = screenSpaceTime;
  times[3] = sampleTime;
  times[4] = compositeTime;
  times[5] = renderTime;

  ifstream dataIn (dfile.c_str());
  if (dataIn.is_open())
  {
    for(int i = 0; i < 7; i++)
    {
        getline (dataIn,line[i]);
        //cout << line[i] << '\n';
    }
    dataIn.close();
  }
  else
  {
    cout << "Unable to open file"<<endl;
    return; 
  }
  ofstream dataOut (dfile.c_str());
  if (dataOut.is_open())
  {
    for(int i = 0; i < 7; i++)
    {
         if(i ==  0) dataOut << line[i] << separator <<numPasses<<endl;
         else dataOut << line[i] << separator <<times[i-1]<<endl;
    }
    
    dataOut.close();
  }
  else dataOut << "Unable to open file";
    string space = " ";

}

//-------------------------------------------------

void  eavlSimpleVRMutator::freeTextures()
{
    if (scalars_array != NULL) 
    {
        scalars_array->unbind(scalars_tref);
        delete scalars_array;
        scalars_array = NULL;
    }

}

//-------------------------------------------------

void  eavlSimpleVRMutator::freeRaw()
{
}

//-------------------------------------------------
void eavlSimpleVRMutator::readTransferFunction(string filename)
{

    std::fstream file(filename.c_str(), std::ios_base::in);
    if(file != NULL)
    {
        //file format number of peg points, then peg points 
        //peg point 0 0 255 255 0.0241845 //RGBA postion(float)
        int numPegs;
        file>>numPegs;
        if(numPegs >= COLOR_MAP_SIZE || numPegs < 1) 
        {
            cerr<<"Invalid number of peg points, valid range [1,1024]: "<<numPegs<<endl;
            exit(1);
        } 

        float *rgb = new float[numPegs*3];
        float *positions = new float[numPegs];
        int trash;
        for(int i = 0; i < numPegs; i++)
        {
            file>>rgb[i*3 + 0];
            file>>rgb[i*3 + 1];
            file>>rgb[i*3 + 2];
            rgb[i*3 + 0] = rgb[i*3 + 0] / 255.f; //normalize
            rgb[i*3 + 1] = rgb[i*3 + 1] / 255.f; //normalize
            rgb[i*3 + 2] = rgb[i*3 + 2] / 255.f; //normalize
            file>>trash;
            file>>positions[i];

        }
        //next we read in the free form opacity
        int numOpacity;
        file>>numOpacity;
        if(numOpacity >= COLOR_MAP_SIZE || numOpacity < 1) 
        {
            cerr<<"Invalid number of opacity points, valid range [1,1024]: "<<numOpacity<<endl;
            exit(1);
        } 
        float *opacityPoints = new float[numOpacity];
        float *opacityPositions = new float[numOpacity];
        cout<<"Num opacity "<<numOpacity<<endl;
        for(int i = 0; i < numOpacity; i++)
        {
            file>>opacityPoints[i];
            cout<<"Opacity point "<<opacityPoints[i]<<endl;
            opacityPoints[i] = (opacityPoints[i] / 255.f ) * opacityFactor; //normalize
            cout<<"Opacity point 2"<<opacityPoints[i]<<endl;
            opacityPositions[i] = i / (float) numOpacity;
        }
        cout<<endl;
        //build the color map

        int rgbPeg1 = 0;
        int rgbPeg2 = 1;

        int opacityPeg1 = 0;
        int opacityPeg2 = 1;
        
        float currentPosition = 0.f;
        float *colorMap = new float[COLOR_MAP_SIZE * 4];

        //fill in rgb values
        float startPosition;
        float endPosition;
        float4 startColor = {0,0,0,0};
        float4 endColor = {0,0,0,0};
        //init color and positions
        if(positions[rgbPeg1] == 0.f)
        {
            startPosition = positions[rgbPeg1];
            startColor.x = rgb[rgbPeg1*3 + 0];
            startColor.y = rgb[rgbPeg1*3 + 1];
            startColor.z = rgb[rgbPeg1*3 + 2];
            endPosition = positions[rgbPeg2];
            endColor.x = rgb[rgbPeg2*3 + 0];
            endColor.y = rgb[rgbPeg2*3 + 1];
            endColor.z = rgb[rgbPeg2*3 + 2];
        }
        else
        {
            //cout<<"init 0 start"<<endl;
            startPosition = 0;
            //color already 0
            endPosition = positions[rgbPeg1];
            endColor.x = rgb[rgbPeg1*3 + 0];
            endColor.y = rgb[rgbPeg1*3 + 1];
            endColor.z = rgb[rgbPeg1*3 + 2];
        }

        for(int i = 0; i < COLOR_MAP_SIZE; i++)
        {
            
            currentPosition = i / (float)COLOR_MAP_SIZE;

            float t = (currentPosition - startPosition) / (endPosition - startPosition);
            colorMap[i*4 + 0] = lerp(startColor.x, endColor.x, t);
            colorMap[i*4 + 1] = lerp(startColor.y, endColor.y, t);
            colorMap[i*4 + 2] = lerp(startColor.z, endColor.z, t);

            if( (currentPosition > endPosition) )
            {
                //advance peg points

                rgbPeg1++;
                rgbPeg2++;  
                //reached the last Peg point 
                if(rgbPeg2 >= numPegs) 
                {
                    startPosition = positions[rgbPeg1];
                    startColor.x = rgb[rgbPeg1*3 + 0];
                    startColor.y = rgb[rgbPeg1*3 + 1];
                    startColor.z = rgb[rgbPeg1*3 + 2];
                    //just keep the same color, we could change this to 0
                    endPosition = 1.f;
                    endColor.x = rgb[rgbPeg1*3 + 0];
                    endColor.y = rgb[rgbPeg1*3 + 1];
                    endColor.z = rgb[rgbPeg1*3 + 2];

                }
                else
                {
                    startPosition = positions[rgbPeg1];
                    startColor.x = rgb[rgbPeg1*3 + 0];
                    startColor.y = rgb[rgbPeg1*3 + 1];
                    startColor.z = rgb[rgbPeg1*3 + 2];
                    endPosition = positions[rgbPeg2];
                    endColor.x = rgb[rgbPeg2*3 + 0];
                    endColor.y = rgb[rgbPeg2*3 + 1];
                    endColor.z = rgb[rgbPeg2*3 + 2];
                }

            }
        }

        float startAlpha = 0.f;
        float endAlpha = 1.f;
        if(positions[opacityPeg1] == 0.f)
        {
            startPosition = opacityPositions[opacityPeg1];
            startAlpha = opacityPoints[opacityPeg1];
            endPosition = opacityPositions[opacityPeg2];
            endAlpha = opacityPoints[opacityPeg2];
        }
        else
        {
            startPosition = 0.f;
            startAlpha = 0.f;
            endPosition = opacityPoints[opacityPeg1];
            endAlpha = opacityPoints[opacityPeg1];
        }
        // fill in alphas
        for(int i = 0; i < COLOR_MAP_SIZE; i++)
        {
           
            currentPosition = i / (float)COLOR_MAP_SIZE;

            float t = (currentPosition - startPosition) / (endPosition - startPosition);
            colorMap[i*4 + 3] = lerp(startAlpha, endAlpha, t);

            //cout<<colorMap[i*4+0]<<" "<<colorMap[i*4+1]<<" "<<colorMap[i*4+2]<<" "<<colorMap[i*4+3]<<" pos "<<currentPosition<<endl;
            if(currentPosition > endPosition)
            {
                //advance peg points

                opacityPeg1++;
                opacityPeg2++;  
                //reached the last Peg point
                if(opacityPeg2 >= numOpacity) 
                {
                    startPosition = opacityPositions[opacityPeg1];
                    startAlpha = opacityPoints[opacityPeg1];
                   
                    //just keep the same color, we could change this to 0
                    endPosition = 1.f;
                    endAlpha = opacityPoints[opacityPeg1];
                    

                }
                else
                {
                    startPosition = opacityPositions[opacityPeg1];
                    startAlpha = opacityPoints[opacityPeg1];
                   
                    endPosition = opacityPositions[opacityPeg2];
                    endAlpha = opacityPoints[opacityPeg2];
                   
                }
            }
        }

        setColorMap4f(colorMap, COLOR_MAP_SIZE);
        delete[] rgb;
        delete[] positions;
        delete[] opacityPoints;
        delete[] opacityPositions;
    }
    else 
    {
        cerr<<"Could not open tranfer function file : "<<filename.c_str()<<endl;
    }
}

eavlByteArray * eavlSimpleVRMutator::getFrameBuffer()
{
    
    eavlExecutor::AddOperation(new_eavlMapOp(eavlOpArgs(framebuffer),
                                             eavlOpArgs(rgba),
                                             CastToUnsignedCharFunctor()),
                                             "set");
    eavlExecutor::Go();
    return rgba;
}
eavlFloatArray * eavlSimpleVRMutator::getRealFrameBuffer()
{
    return framebuffer;
}
void eavlSimpleVRMutator::setSampleLCFlag(int val)
{
    sampleLCFlag = val; 
    cerr<<"sampleLCFlag value changes to "<<sampleLCFlag<<"\n";
}

void eavlSimpleVRMutator::getImageSubsetDims(int *dims)
{
  dims[0] = xmin;
  dims[1] = ymin;
  dims[2] = dx;
  dims[3] = dy;

}
