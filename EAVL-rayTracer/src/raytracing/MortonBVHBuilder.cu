#include "hip/hip_runtime.h"
#include "MortonBVHBuilder.h"
#include "eavlTextureObject.h"
#include "eavlReduceOp_1.h"
#include "eavlMapOp.h"
#include "eavlRadixSortOp.h"
#include "eavlCountingIterator.h"
#include "eavlGatherOp.h"
#include <algorithm>    
using namespace std; 

#ifdef HAVE_OPENMP
#include <omp.h>
#endif

//This is the structure of the flat BVH inner 
//layout
struct FlatIndxr
{
    eavlArrayIndexer xmin1;
    eavlArrayIndexer ymin1;
    eavlArrayIndexer zmin1;
    eavlArrayIndexer xmax1;
    eavlArrayIndexer ymax1;
    eavlArrayIndexer zmax1;
    eavlArrayIndexer xmin2;
    eavlArrayIndexer ymin2;
    eavlArrayIndexer zmin2;
    eavlArrayIndexer xmax2;
    eavlArrayIndexer ymax2;
    eavlArrayIndexer zmax2;

    eavlArrayIndexer lChild;
    eavlArrayIndexer rChild;

    FlatIndxr()
    {
        xmin1.add = 0;
        ymin1.add = 1;
        zmin1.add = 2;
        xmax1.add = 3;
        ymax1.add = 4;
        zmax1.add = 5;
        xmin2.add = 6;
        ymin2.add = 7;
        zmin2.add = 8;
        xmax2.add = 9;
        ymax2.add = 10;
        zmax2.add = 11;

        lChild.add = 12;
        rChild.add = 13;

        xmin1.mul = 16;
        ymin1.mul = 16;
        zmin1.mul = 16;
        xmax1.mul = 16;
        ymax1.mul = 16;
        zmax1.mul = 16;
        xmin2.mul = 16;
        ymin2.mul = 16;
        zmin2.mul = 16;
        xmax2.mul = 16;
        ymax2.mul = 16;
        zmax2.mul = 16;

        lChild.mul = 16;
        rChild.mul = 16;
        //two extra floats with padding not shown
    }
};

void validate(BVHSOA *bvh, int numLeafs, int currentNode, int &count)
{
    if(currentNode >= numLeafs - 1) 
    {
        count++;
        return; //at leaf: inc and get out
    }

    validate(bvh, numLeafs, bvh->leftChild->GetValue(currentNode), count);
    validate(bvh, numLeafs, bvh->rightChild->GetValue(currentNode), count);

}


MortonBVHBuilder::MortonBVHBuilder(eavlFloatArray* _verts, int _numPrimitives, primitive_t _primitveType)
  : verts(_verts), numPrimitives(_numPrimitives), primitveType(_primitveType)
{

      verbose = 0;
    
      convertedToAoS = false;
      wasEavlArrayGiven = false;
      if(numPrimitives < 1) THROW(eavlException, "Number of primitives must be greater that zero.");
      if(verts == NULL)     THROW(eavlException, "Verticies can't be NULL");
      //Insert preprocess that splits triangles before any of the memory is allocated
      bvh     = new BVHSOA(numPrimitives);
      indexes = new eavlIntArray("idx",1,numPrimitives);
      tmpInt  = new eavlIntArray("tmp",1,numPrimitives);
    
      eavlCountingIterator::generateIterator(indexes);

      mortonCodes = new eavlIntArray("mortonCodes",1,numPrimitives);

      tmpFloat   = new eavlFloatArray("tmpSpace",1, 2 * numPrimitives -1);
}

MortonBVHBuilder::~MortonBVHBuilder()
{
    delete mortonCodes;
    delete bvh;
    delete indexes;
    delete tmpFloat;
    delete tmpInt;
    if(!wasEavlArrayGiven) //we gave up control of this memory
    {
        delete leafNodes;
        delete innerNodes;    
    }
    
}
//TODO: this might work better with Global mem since only a few values are accessed


template<primitive_t primType> 
struct AABBFunctor
{ 
	eavlTextureObject<float4> verts;
	AABBFunctor(eavlTextureObject<float4> *_verts)
	: verts(*_verts)
	{}
	EAVL_FUNCTOR tuple<float, float, float, float, float, float> operator()(int idx)
	{
        float xmin;
        float xmax;
        float ymin;
        float ymax;
        float zmin;
        float zmax;

        if(primType == SPHERE)
        {
            float4 sdata = verts.getValue(idx);
            eavlVector3 temp(0,0,0);
            eavlVector3 center( sdata.x, sdata.y, sdata.z );
            
            float radius = sdata.w;
            temp.x = radius;
            temp.y = 0;
            temp.z = 0;

            eavlVector3 p = center + temp;
            //set first point to max and min
            xmin = p.x; xmax = p.x;
            ymin = p.y; ymax = p.y;
            zmin = p.z; zmax = p.z;

            p = center - temp;
            xmin = min(xmin, p.x); xmax = max(xmax, p.x);
            ymin = min(ymin, p.y); ymax = max(ymax, p.y);
            zmin = min(zmin, p.z); zmax = max(zmax, p.z);
            
            temp.x = 0;
            temp.y = radius;
            temp.z = 0;
            
            p = center + temp;
            xmin = min(xmin, p.x); xmax = max(xmax, p.x);
            ymin = min(ymin, p.y); ymax = max(ymax, p.y);
            zmin = min(zmin, p.z); zmax = max(zmax, p.z);

            p = center - temp;
            xmin = min(xmin, p.x); xmax = max(xmax, p.x);
            ymin = min(ymin, p.y); ymax = max(ymax, p.y);
            zmin = min(zmin, p.z); zmax = max(zmax, p.z);

            temp.x = 0;
            temp.y = 0;
            temp.z = radius;
            p = center + temp;
            xmin = min(xmin, p.x); xmax = max(xmax, p.x);
            ymin = min(ymin, p.y); ymax = max(ymax, p.y);
            zmin = min(zmin, p.z); zmax = max(zmax, p.z);

            p = center - temp;
            xmin = min(xmin, p.x); xmax = max(xmax, p.x);
            ymin = min(ymin, p.y); ymax = max(ymax, p.y);
            zmin = min(zmin, p.z); zmax = max(zmax, p.z);
        }

        if(primType == CYLINDER)
        {
            eavlVector3 temp(0,0,0);
            float4 c1 = verts.getValue(idx * 2);
            float4 c2 = verts.getValue(idx * 2 + 1);
            eavlVector3 base( c1.x, c1.y, c1.z );
            float radius = c1.w;
            eavlVector3 axis( c2.x, c2.y, c2.z );
            float height = c2.w;
            eavlVector3 top = base + axis * height;

            
            temp.x = radius;
            temp.y = 0;
            temp.z = 0;

            eavlVector3 p = base + temp;
            xmin = p.x; xmax = p.x;
            ymin = p.y; ymax = p.y;
            zmin = p.z; zmax = p.z;

            p = base - temp;
            xmin = min(xmin, p.x); xmax = max(xmax, p.x);
            ymin = min(ymin, p.y); ymax = max(ymax, p.y);
            zmin = min(zmin, p.z); zmax = max(zmax, p.z);

            p = top + temp;
            xmin = min(xmin, p.x); xmax = max(xmax, p.x);
            ymin = min(ymin, p.y); ymax = max(ymax, p.y);
            zmin = min(zmin, p.z); zmax = max(zmax, p.z);

            p = top - temp;
            xmin = min(xmin, p.x); xmax = max(xmax, p.x);
            ymin = min(ymin, p.y); ymax = max(ymax, p.y);
            zmin = min(zmin, p.z); zmax = max(zmax, p.z);

           
            temp.x = 0;
            temp.y = radius;
            temp.z = 0;
            p = base + temp;
            xmin = min(xmin, p.x); xmax = max(xmax, p.x);
            ymin = min(ymin, p.y); ymax = max(ymax, p.y);
            zmin = min(zmin, p.z); zmax = max(zmax, p.z);

            p = base - temp;
            xmin = min(xmin, p.x); xmax = max(xmax, p.x);
            ymin = min(ymin, p.y); ymax = max(ymax, p.y);
            zmin = min(zmin, p.z); zmax = max(zmax, p.z);

            p = top + temp;
            xmin = min(xmin, p.x); xmax = max(xmax, p.x);
            ymin = min(ymin, p.y); ymax = max(ymax, p.y);
            zmin = min(zmin, p.z); zmax = max(zmax, p.z);

            p = top - temp;
            xmin = min(xmin, p.x); xmax = max(xmax, p.x);
            ymin = min(ymin, p.y); ymax = max(ymax, p.y);
            zmin = min(zmin, p.z); zmax = max(zmax, p.z);

            temp.x = 0;
            temp.y = 0;
            temp.z = radius;
            
            p = base + temp;
            xmin = min(xmin, p.x); xmax = max(xmax, p.x);
            ymin = min(ymin, p.y); ymax = max(ymax, p.y);
            zmin = min(zmin, p.z); zmax = max(zmax, p.z);
            
            p = base - temp;
            xmin = min(xmin, p.x); xmax = max(xmax, p.x);
            ymin = min(ymin, p.y); ymax = max(ymax, p.y);
            zmin = min(zmin, p.z); zmax = max(zmax, p.z);

            p = top + temp;
            xmin = min(xmin, p.x); xmax = max(xmax, p.x);
            ymin = min(ymin, p.y); ymax = max(ymax, p.y);
            zmin = min(zmin, p.z); zmax = max(zmax, p.z);

            p = top - temp;
            xmin = min(xmin, p.x); xmax = max(xmax, p.x);
            ymin = min(ymin, p.y); ymax = max(ymax, p.y);
            zmin = min(zmin, p.z); zmax = max(zmax, p.z);
        }
    	

        return tuple<float, float, float, float, float, float>(xmin, ymin, zmin, xmax, ymax, zmax);
	} 
};


struct AABBTriFunctor
{ 
    eavlTextureObject<float> verts;
    AABBTriFunctor(eavlTextureObject<float> *_verts)
    : verts(*_verts)
    {}
    EAVL_FUNCTOR tuple<float, float, float, float, float, float> operator()(int idx)
    {
        float xmin;
        float xmax;
        float ymin;
        float ymax;
        float zmin;
        float zmax;

        eavlVector3 a,b,c;
        a.x = verts.getValue(idx * 9 + 0);
        a.y = verts.getValue(idx * 9 + 1);
        a.z = verts.getValue(idx * 9 + 2);
        b.x = verts.getValue(idx * 9 + 3);
        b.y = verts.getValue(idx * 9 + 4);
        b.z = verts.getValue(idx * 9 + 5);
        c.x = verts.getValue(idx * 9 + 6);
        c.y = verts.getValue(idx * 9 + 7);
        c.z = verts.getValue(idx * 9 + 8);

        xmin = min(a.x, min(b.x, c.x));
        xmax = max(a.x, max(b.x, c.x));
        ymin = min(a.y, min(b.y, c.y));
        ymax = max(a.y, max(b.y, c.y));
        zmin = min(a.z, min(b.z, c.z));
        zmax = max(a.z, max(b.z, c.z));

        return tuple<float, float, float, float, float, float>(xmin, ymin, zmin, xmax, ymax, zmax);
    } 
};

struct CentroidFunctor
{

	CentroidFunctor(){}
	EAVL_FUNCTOR tuple<float, float, float> operator()(tuple<float, float, float, float, float, float>  bbox)
	{												
		eavlVector3 minPoint(get<0>(bbox),get<1>(bbox),get<2>(bbox));
		eavlVector3 maxPoint(get<3>(bbox),get<4>(bbox),get<5>(bbox));

		eavlVector3 dir = maxPoint - minPoint;

		float halfDist = sqrt(dir * dir) * 0.5f;

		dir.normalize();

		eavlVector3 centroid = minPoint + halfDist * dir;  

        return tuple<float, float, float>(centroid.x, centroid.y, centroid.z);
	} 
};

struct MortonFunctor
{
	eavlVector3 mins;
	eavlVector3 invExtent;
	MortonFunctor(const eavlVector3 &mn, const eavlVector3 &mx)
	: mins(mn)
	{
		invExtent = mx - mn;
		invExtent.x = (invExtent.x == 0) ? 0 : 1.f / invExtent.x;
		invExtent.y = (invExtent.y == 0) ? 0 : 1.f / invExtent.y;
		invExtent.z = (invExtent.z == 0) ? 0 : 1.f / invExtent.z;
	}
	EAVL_FUNCTOR tuple<int> operator()(tuple<float, float, float>  input)
	{												
		eavlVector3 centroid(get<0>(input),get<1>(input),get<2>(input));
		//normalize to the unit cube
		centroid -= mins;
		centroid.x = centroid.x * invExtent.x;
		centroid.y = centroid.y * invExtent.y;
		centroid.z = centroid.z * invExtent.z;

		unsigned int code = morton3D(centroid.x, centroid.y, centroid.z); 
        return tuple<int>(code);
	} 
};

struct TreeFunctor
{ 
	int leafCount;
    int innerCount;

	eavlTextureObject<unsigned int> mortonCodes;
    eavlFunctorArray<int> parents;


	TreeFunctor(eavlTextureObject<unsigned int> *codes, 
                int _leafCount, 
                eavlFunctorArray<int> par)
	: mortonCodes(*codes), leafCount(_leafCount), parents(par)
	{
        innerCount = leafCount - 1;
    }

	/**
	 * returns the count of largest shared prefix between
	 * two morton codes. Ties are broken by the indexes
	 * a and b.
	 * @param  a - index of value one
	 * @param  b - index of value two
	 * @return count of the largest binary prefix 
	 */
	EAVL_HOSTDEVICE int cclz(unsigned int &x)
	{
	  unsigned int y;
	  int n = 32;
	  y = x >>16; if (y != 0) { n = n -16; x = y; }
    y = x >> 8; if (y != 0) { n = n - 8; x = y; }
    y = x >> 4; if (y != 0) { n = n - 4; x = y; }
    y = x >> 2; if (y != 0) { n = n - 2; x = y; }
    y = x >> 1; if (y != 0) return n - 2;
    return n - x;
	}
	EAVL_HOSTDEVICE int delta(const int &a, const int &b)
	{
		bool tie = false;
		bool outOfRange = (b < 0 || b > leafCount -1);
        int bb = (outOfRange) ? 0 : b; //still make the call but with a valid adderss
		unsigned int aCode =  mortonCodes.getValue(a);
		unsigned int bCode =  mortonCodes.getValue(bb);
		unsigned int exOr = aCode ^ bCode; //use xor to find where they differ
		tie = (exOr == 0);
		exOr = tie ? a ^ bb : exOr; //break the tie, a and b will always differ 
#ifdef __CUDA_ARCH__
    int count = clz(exOr);
#else 
    //
    //  Aparently on Surface@llnl this doesn't work.
    //
    int count = cclz(exOr);
#endif
		if(tie) count += 32; 
		count = (outOfRange) ? -1 : count;
		return count;
	}

	EAVL_FUNCTOR tuple<int, int> operator()(int idx)
	{								
        if(idx > leafCount - 2) return tuple<int, int>(-1,-1);					
        //determine range direction
        int d = 0 > (delta(idx, idx + 1) - delta(idx, idx - 1)) ?  -1 : 1;
        
        //find upper bound for the length of the range
        int minDelta = delta(idx, idx - d);
        int lMax = 2;
        while( delta(idx, idx + lMax * d) > minDelta ) lMax *= 2; 

        //binary search to find the lower bound
        int l = 0;
        for(int t = lMax / 2; t >= 1; t/=2) 
        {
            if(delta(idx, idx + (l + t)*d ) > minDelta) l += t;
        }

        int j = idx + l * d;
        int deltaNode = delta(idx,j);
        int s = 0;
        float divFactor = 2.f; 
        //find the split postition using a binary search
        for(int t = (int) ceil(l / divFactor);; divFactor*=2, t = (int) ceil(l / divFactor) )
        {    
            if(delta(idx, idx + (s + t) * d) > deltaNode)
            {
                s += t;
            } 
            
            if(t == 1) break;
        }

        int split = idx + s * d + min(d,0);
        int leftChild, rightChild;
        //assign parent/child pointers
        if(min(idx, j) == split)
        {
          //leaf
          parents[split + innerCount] = idx;
          leftChild = split + innerCount;
        } 
        else
        {
            parents[split] = idx;
            leftChild = split;        //inner node   
        }
         

        if(max(idx, j) == split + 1)
        {
           //leaf
          parents[split + innerCount + 1] = idx;
          rightChild = split + innerCount + 1;  
        } 
        else
        {
            parents[split + 1] = idx;
            rightChild = split + 1;       
        }
        return tuple<int, int>(leftChild,rightChild);
	} 
};

struct BottomUpFunctor
{
    eavlFunctorArray<int> nodeCounters;
    int numLeafs;
    eavlFunctorArray<float> xmins;
    eavlFunctorArray<float> ymins;
    eavlFunctorArray<float> zmins;
    eavlFunctorArray<float> xmaxs;
    eavlFunctorArray<float> ymaxs;
    eavlFunctorArray<float> zmaxs;
    eavlFunctorArray<int>   lChild;
    eavlFunctorArray<int>   rChild;
    eavlFunctorArray<int>   parents;

    BottomUpFunctor(eavlFunctorArray<float> _xmins, 
                    eavlFunctorArray<float> _ymins, 
                    eavlFunctorArray<float> _zmins,
                    eavlFunctorArray<float> _xmaxs, 
                    eavlFunctorArray<float> _ymaxs, 
                    eavlFunctorArray<float> _zmaxs,
                    eavlFunctorArray<int> _lChild, 
                    eavlFunctorArray<int> _rChild,
                    eavlFunctorArray<int> _parents,
                    eavlFunctorArray<int> aCounters, int _numLeafs)
    :  xmins(_xmins), ymins(_ymins), zmins(_zmins),
       xmaxs(_xmaxs), ymaxs(_ymaxs), zmaxs(_zmaxs),
       lChild(_lChild), rChild(_rChild), parents(_parents),
       nodeCounters(aCounters), numLeafs(_numLeafs)
    {
        
    }

    EAVL_HOSTDEVICE bool checkIdx(int idx)
    {
        //if(idx < 0 || idx >= (numLeafs - 1 )) printf("ILLEGAL %d ", idx);
        int old;
        bool kill = false;
#ifdef __CUDA_ARCH__
        old = atomicAdd(&nodeCounters[idx],1);   
#else 
        #pragma omp atomic capture
        old = nodeCounters[idx]++;
#endif
        if(old == 0) kill = true;
        // if(kill) printf("Idx %d \n", idx);
        return kill;
    }


    EAVL_FUNCTOR tuple<int> operator()(int idx)
    {   
        //start traversal at the leaves                                            
        int node = idx + numLeafs - 1;
        do
        {
            //Go up one level to make sure everyone has children
            node = parents[node];
            //First thread to get to the node terminates,
            //Second  thread processes AABB of the children
            if(checkIdx(node)) return tuple<int>(idx);
            xmins[node] = min(xmins[lChild[node]], xmins[rChild[node]]);
            ymins[node] = min(ymins[lChild[node]], ymins[rChild[node]]);
            zmins[node] = min(zmins[lChild[node]], zmins[rChild[node]]);
            xmaxs[node] = max(xmaxs[lChild[node]], xmaxs[rChild[node]]);
            ymaxs[node] = max(ymaxs[lChild[node]], ymaxs[rChild[node]]);
            zmaxs[node] = max(zmaxs[lChild[node]], zmaxs[rChild[node]]);
        } while (node != 0);
        
        ;//printf("There can only be one! %d\n", idx);

        return tuple<int>(idx); //indexed to same value / Do nothing
    } 
};


struct InnerToFlatFunctor
{
    eavlFunctorArray<float> xmins;
    eavlFunctorArray<float> ymins;
    eavlFunctorArray<float> zmins;
    eavlFunctorArray<float> xmaxs;
    eavlFunctorArray<float> ymaxs;
    eavlFunctorArray<float> zmaxs;
    int    numPrimitives;
    int    primOffset;
    InnerToFlatFunctor(eavlFunctorArray<float> _xmins, 
                       eavlFunctorArray<float> _ymins, 
                       eavlFunctorArray<float> _zmins,
                       eavlFunctorArray<float> _xmaxs, 
                       eavlFunctorArray<float> _ymaxs, 
                       eavlFunctorArray<float> _zmaxs,
                       int _numPrims) 
                       : xmins(_xmins), ymins(_ymins), zmins(_zmins),
                         xmaxs(_xmaxs), ymaxs(_ymaxs), zmaxs(_zmaxs),
                         numPrimitives(_numPrims)
    {
        primOffset = numPrimitives - 1;
    }
    EAVL_FUNCTOR   tuple<float, float, float, float, float, float, float, float, float, float, float, float, float, float>
        operator()(tuple<int, int>  input)
    {                                               
        int lChild = get<0>(input);
        int rChild = get<1>(input);
        //Child nodes are indexed with neg numbers in this format
        //First prim is at -1
        int clIdx  = (lChild < numPrimitives - 1) ? lChild * 4 : -(lChild - primOffset) * 2 - 1;  
        int crIdx  = (rChild < numPrimitives - 1) ? rChild * 4 : -(rChild - primOffset) * 2 - 1;  
        //to avoid truncating interger value when casting to floating point
        float clf,crf;
        memcpy(&clf,&clIdx, 4);
        memcpy(&crf,&crIdx, 4);
        //each node stores the bounding boxes of its children 
        return tuple<float, float, float, float, float, float, float, float, float, float, float, float, float, float>
        (xmins[lChild], ymins[lChild], zmins[lChild], xmaxs[lChild], ymaxs[lChild], zmaxs[lChild],
         xmins[rChild], ymins[rChild], zmins[rChild], xmaxs[rChild], ymaxs[rChild], zmaxs[rChild],
         clf, crf);
    } 
};


struct LeafToFlatFunctor
{
    LeafToFlatFunctor(){}
    EAVL_FUNCTOR tuple<int, int> operator()(int  primId)
    {                                               
        return tuple<int, int>(1, primId);
    } 
};


void MortonBVHBuilder::setVerbose(const int &level)
{
	if(level > 0) verbose = level;
}

void MortonBVHBuilder::findAABBs()
{
    eavlTextureObject<float> *floatVerts = NULL;
    eavlTextureObject<float4> *float4Verts = NULL;
    //load verts into texture for bbox calculation
    //TODO:Does this make sense to have this as texture? 
    //3 reads per thread not really streaming many addresses
    if(primitveType == TRIANGLE)
    {
        floatVerts = new eavlTextureObject<float>(numPrimitives * 9, verts, false);
    }
    else if(primitveType == SPHERE)
    {
        float4Verts = new eavlTextureObject<float4>(numPrimitives, (float4*)verts, false);
    }
    else if(primitveType == CYLINDER)
    {
        float4Verts = new eavlTextureObject<float4>(numPrimitives * 2, (float4*)verts, false);
    }
    

	//calculate the AABBs of all the primitives
    if(primitveType == TRIANGLE)
    {
        eavlExecutor::AddOperation(
        new_eavlMapOp(eavlOpArgs(indexes),
                      eavlOpArgs(eavlIndexable<eavlFloatArray>(bvh->xmin, *bvh->leafIndexer),
                                 eavlIndexable<eavlFloatArray>(bvh->ymin, *bvh->leafIndexer),
                                 eavlIndexable<eavlFloatArray>(bvh->zmin, *bvh->leafIndexer),
                                 eavlIndexable<eavlFloatArray>(bvh->xmax, *bvh->leafIndexer),
                                 eavlIndexable<eavlFloatArray>(bvh->ymax, *bvh->leafIndexer),
                                 eavlIndexable<eavlFloatArray>(bvh->zmax, *bvh->leafIndexer)),
                      AABBTriFunctor(floatVerts)),
                      "AABB");
        eavlExecutor::Go();
    }
    else if(primitveType == SPHERE)
    {
        eavlExecutor::AddOperation(
        new_eavlMapOp(eavlOpArgs(indexes),
                      eavlOpArgs(eavlIndexable<eavlFloatArray>(bvh->xmin, *bvh->leafIndexer),
                                 eavlIndexable<eavlFloatArray>(bvh->ymin, *bvh->leafIndexer),
                                 eavlIndexable<eavlFloatArray>(bvh->zmin, *bvh->leafIndexer),
                                 eavlIndexable<eavlFloatArray>(bvh->xmax, *bvh->leafIndexer),
                                 eavlIndexable<eavlFloatArray>(bvh->ymax, *bvh->leafIndexer),
                                 eavlIndexable<eavlFloatArray>(bvh->zmax, *bvh->leafIndexer)),
                      AABBFunctor<SPHERE>(float4Verts)),
                      "AABB");
        eavlExecutor::Go();
    }
    else if(primitveType == CYLINDER)
    {
        eavlExecutor::AddOperation(
        new_eavlMapOp(eavlOpArgs(indexes),
                      eavlOpArgs(eavlIndexable<eavlFloatArray>(bvh->xmin, *bvh->leafIndexer),
                                 eavlIndexable<eavlFloatArray>(bvh->ymin, *bvh->leafIndexer),
                                 eavlIndexable<eavlFloatArray>(bvh->zmin, *bvh->leafIndexer),
                                 eavlIndexable<eavlFloatArray>(bvh->xmax, *bvh->leafIndexer),
                                 eavlIndexable<eavlFloatArray>(bvh->ymax, *bvh->leafIndexer),
                                 eavlIndexable<eavlFloatArray>(bvh->zmax, *bvh->leafIndexer)),
                      AABBFunctor<CYLINDER>(float4Verts)),
                      "AABB");
        eavlExecutor::Go();
    }
	
    if(floatVerts  != NULL) delete floatVerts;
    if(float4Verts != NULL) delete float4Verts;

    //we have to create the legacy indexer since Reduce is old.
    //It will also never be updated.
    eavlArrayWithLinearIndex lIndexer;
    lIndexer.div = 1;
    lIndexer.mul = 1;
    lIndexer.mod = INT_MAX;
    lIndexer.add = numPrimitives - 1; //leaf offset
    lIndexer.array = bvh->xmin;


    //create an array to store the reduction into
    eavlFloatArray *value = new eavlFloatArray("",1,1);
    //find the min and the max extents for each coordinate;
    //min
    eavlExecutor::AddOperation(
        new eavlReduceOp_1<eavlMinFunctor<float> >
        (lIndexer, value, eavlMinFunctor<float>(), numPrimitives), "min");

    eavlExecutor::Go();

    bvh->extentMin.x = value->GetValue(0);
    lIndexer.array = bvh->ymin;
    eavlExecutor::AddOperation(
        new eavlReduceOp_1<eavlMinFunctor<float> >
        (lIndexer, value, eavlMinFunctor<float>(), numPrimitives), "min");

    eavlExecutor::Go();

    bvh->extentMin.y = value->GetValue(0);
    lIndexer.array = bvh->zmin;
    eavlExecutor::AddOperation(
        new eavlReduceOp_1<eavlMinFunctor<float> >
        (lIndexer, value, eavlMinFunctor<float>(), numPrimitives), "min");

    eavlExecutor::Go();

    bvh->extentMin.z = value->GetValue(0);
    //max
    lIndexer.array = bvh->xmax;
    eavlExecutor::AddOperation(
        new eavlReduceOp_1<eavlMaxFunctor<float> >
        (lIndexer, value, eavlMaxFunctor<float>(), numPrimitives), "max");

    eavlExecutor::Go();

    bvh->extentMax.x = value->GetValue(0);
    lIndexer.array = bvh->ymax;
    eavlExecutor::AddOperation(
        new eavlReduceOp_1<eavlMaxFunctor<float> >
        (lIndexer, value, eavlMaxFunctor<float>(), numPrimitives), "max");

    eavlExecutor::Go();

    bvh->extentMax.y = value->GetValue(0);
    lIndexer.array = bvh->zmax;
    eavlExecutor::AddOperation(
        new eavlReduceOp_1<eavlMaxFunctor<float> >
        (lIndexer, value, eavlMaxFunctor<float>(), numPrimitives), "max");

    eavlExecutor::Go();

    bvh->extentMax.z = value->GetValue(0);

    delete value;
    if(verbose > 0) bvh->print();

    eavlExecutor::AddOperation(
    	new_eavlMapOp(eavlOpArgs(eavlIndexable<eavlFloatArray>(bvh->xmin, *bvh->leafIndexer),
                                 eavlIndexable<eavlFloatArray>(bvh->ymin, *bvh->leafIndexer),
                                 eavlIndexable<eavlFloatArray>(bvh->zmin, *bvh->leafIndexer),
                                 eavlIndexable<eavlFloatArray>(bvh->xmax, *bvh->leafIndexer),
                                 eavlIndexable<eavlFloatArray>(bvh->ymax, *bvh->leafIndexer),
                                 eavlIndexable<eavlFloatArray>(bvh->zmax, *bvh->leafIndexer)),
                      eavlOpArgs(bvh->centroidX, bvh->centroidY, bvh->centroidZ), 
                      CentroidFunctor(), numPrimitives ),
                      "Centroid");
    eavlExecutor::Go();
}

void MortonBVHBuilder::sort()
{
    // primitive ids and scatter indexes are the same
	eavlIntArray *idx = bvh->primId;
    eavlExecutor::AddOperation(
		new_eavlRadixSortOp(eavlOpArgs(mortonCodes),
                            eavlOpArgs(idx), true),
                            "Radix");
    eavlExecutor::Go();
    int tgather;

    /**
     * Allocation takes a long time, so 
     * we keep a temp array around to avoid 
     * that cost. Just do some pointer swapping.
     */
    eavlFloatArray *tmpPtr = NULL;
    if(verbose > 0) tgather = eavlTimer::Start();

    eavlExecutor::AddOperation(
        new_eavlGatherOp(eavlOpArgs(eavlIndexable<eavlFloatArray>(bvh->xmin, *bvh->leafIndexer)),
                         eavlOpArgs(eavlIndexable<eavlFloatArray>(tmpFloat,  *bvh->leafIndexer)),
                         eavlOpArgs(idx), numPrimitives),
                         "sorting");
    eavlExecutor::Go();
    tmpPtr = bvh->xmin;
    bvh->xmin = tmpFloat;
    tmpFloat = tmpPtr;

    eavlExecutor::AddOperation(
        new_eavlGatherOp(eavlOpArgs(eavlIndexable<eavlFloatArray>(bvh->ymin, *bvh->leafIndexer)),
                         eavlOpArgs(eavlIndexable<eavlFloatArray>(tmpFloat,  *bvh->leafIndexer)),
                         eavlOpArgs(idx), numPrimitives),
                         "sorting");
    eavlExecutor::Go();
    tmpPtr = bvh->ymin;
    bvh->ymin = tmpFloat;
    tmpFloat = tmpPtr;

    eavlExecutor::AddOperation(
        new_eavlGatherOp(eavlOpArgs(eavlIndexable<eavlFloatArray>(bvh->zmin, *bvh->leafIndexer)),
                         eavlOpArgs(eavlIndexable<eavlFloatArray>(tmpFloat,  *bvh->leafIndexer)),
                         eavlOpArgs(idx), numPrimitives),
                         "sorting");
    eavlExecutor::Go();
    tmpPtr = bvh->zmin;
    bvh->zmin = tmpFloat;
    tmpFloat = tmpPtr;

    eavlExecutor::AddOperation(
        new_eavlGatherOp(eavlOpArgs(eavlIndexable<eavlFloatArray>(bvh->xmax, *bvh->leafIndexer)),
                         eavlOpArgs(eavlIndexable<eavlFloatArray>(tmpFloat,  *bvh->leafIndexer)),
                         eavlOpArgs(idx), numPrimitives),
                         "sorting");
    eavlExecutor::Go();
    tmpPtr = bvh->xmax;
    bvh->xmax = tmpFloat;
    tmpFloat = tmpPtr;

    eavlExecutor::AddOperation(
        new_eavlGatherOp(eavlOpArgs(eavlIndexable<eavlFloatArray>(bvh->ymax, *bvh->leafIndexer)),
                         eavlOpArgs(eavlIndexable<eavlFloatArray>(tmpFloat,  *bvh->leafIndexer)),
                         eavlOpArgs(idx), numPrimitives),
                         "sorting");
    eavlExecutor::Go();
    tmpPtr = bvh->ymax;
    bvh->ymax = tmpFloat;
    tmpFloat = tmpPtr;

    eavlExecutor::AddOperation(
        new_eavlGatherOp(eavlOpArgs(eavlIndexable<eavlFloatArray>(bvh->zmax, *bvh->leafIndexer)),
                         eavlOpArgs(eavlIndexable<eavlFloatArray>(tmpFloat,  *bvh->leafIndexer)),
                         eavlOpArgs(idx), numPrimitives),
                         "sorting");
    eavlExecutor::Go();
    tmpPtr = bvh->zmax;
    bvh->zmax = tmpFloat;
    tmpFloat = tmpPtr;

    if(verbose > 0) cout<<"GATHER   RUNTIME: "<<eavlTimer::Stop(tgather,"rf")<<endl;
}

void MortonBVHBuilder::propagateAABBs()
{
    eavlExecutor::AddOperation(
        new_eavlMapOp(eavlOpArgs(mortonCodes),
                      eavlOpArgs(mortonCodes),
                      IntMemsetFunctor(0)), "");
    eavlExecutor::Go();

    eavlFunctorArray<int>   atomicCounters(mortonCodes);
    eavlFunctorArray<int>   parents(bvh->parent);
    eavlFunctorArray<int>   lChild(bvh->leftChild);
    eavlFunctorArray<int>   rChild(bvh->rightChild);

    eavlFunctorArray<float> xmins(bvh->xmin);
    eavlFunctorArray<float> ymins(bvh->ymin);
    eavlFunctorArray<float> zmins(bvh->zmin);
    eavlFunctorArray<float> xmaxs(bvh->xmax);
    eavlFunctorArray<float> ymaxs(bvh->ymax);
    eavlFunctorArray<float> zmaxs(bvh->zmax);

    eavlExecutor::AddOperation(
        new_eavlMapOp(eavlOpArgs(indexes),
                      eavlOpArgs(indexes), //dummy
                      BottomUpFunctor(xmins, ymins, zmins,
                                      xmaxs, ymaxs, zmaxs,
                                      lChild, rChild, parents,
                                      atomicCounters, numPrimitives)), "");
    eavlExecutor::Go();

}

void MortonBVHBuilder::build()
{

	int taabb;
    if(verbose > 0) taabb = eavlTimer::Start();
    //Calculate AABBs and centroids of the primitves
	findAABBs();
	if(verbose > 0) cout<<"AABB     RUNTIME: "<<eavlTimer::Stop(taabb,"rf")<<endl;
     //cout<<"VVVVVV "<<bvh->xmin->GetValue(bvh->numInner)<< " "<<bvh->zmax->GetValue(bvh->numInner)<<endl;

	//Generate Morton code based on the centriod of the AABB
	eavlExecutor::AddOperation(
    	new_eavlMapOp(eavlOpArgs(bvh->centroidX, bvh->centroidY, bvh->centroidZ),
                      eavlOpArgs(mortonCodes),
                      MortonFunctor(bvh->extentMin, bvh->extentMax)),
                      "Morton");
    eavlExecutor::Go();
     //cout<<"VVVVVV "<<bvh->xmin->GetValue(bvh->numInner)<< " "<<bvh->zmax->GetValue(bvh->numInner)<<endl;
    int tsort;
    if(verbose > 0) tsort = eavlTimer::Start();
    sort();
    if(verbose > 0) cout<<"SORT     RUNTIME: "<<eavlTimer::Stop(tsort,"rf")<<endl;

    eavlTextureObject<unsigned int> *mortonTexture = NULL;
    mortonTexture = new eavlTextureObject<unsigned int>( numPrimitives, 
                                                         mortonCodes,
                                                         false);
    //cout<<"VVVVVV "<<bvh->xmin->GetValue(bvh->numInner)<< " "<<bvh->zmax->GetValue(bvh->numInner)<<endl;
    //bvh->parent->SetValue(0,-1);
    eavlFunctorArray<int> parents(bvh->parent);

    //Build the tree in place. TODO: figure out a better way to set parent pointers
    // Current method will fail if the GPU falls back to the CPU
    int ttree;
    if(verbose > 0) ttree = eavlTimer::Start();
    eavlExecutor::AddOperation(
        new_eavlMapOp(eavlOpArgs(indexes),
                      eavlOpArgs(bvh->leftChild, bvh->rightChild),
                      TreeFunctor(mortonTexture, numPrimitives,parents), numPrimitives - 1),
                      "tree");
    eavlExecutor::Go();
    if(verbose > 0) cout<<"TREE     RUNTIME: "<<eavlTimer::Stop(ttree,"rf")<<endl;
    int tprop;
    if(verbose > 0) tprop = eavlTimer::Start();
    propagateAABBs();
    if(verbose > 0) cout<<"PROP     RUNTIME: "<<eavlTimer::Stop(tprop,"rf")<<endl;
    
    delete mortonTexture;
    //int count = 0;
    //validate(bvh, numPrimitives, 0, count);
    //if(count != numPrimitives) cout<<"BBBBBBBAADD "<<count<<endl;
    //cout<<bvh->xmin->GetValue(0)<<" "<<bvh->xmax->GetValue(0)<<endl;
}

void MortonBVHBuilder::flatten()
{
    //hand these arrays off to the consumer and let them deaL with deleting them.
    innerNodes = new eavlFloatArray("inner",1, (numPrimitives -1) * 16);  //16 flat values per node
    leafNodes  = new eavlIntArray("leafs",1, numPrimitives * 2);

    eavlFunctorArray<int>   atomicCounters(mortonCodes);
    eavlFunctorArray<int>   parents(bvh->parent);
    eavlFunctorArray<int>   lChild(bvh->leftChild);
    eavlFunctorArray<int>   rChild(bvh->rightChild);

    eavlFunctorArray<float> xmins(bvh->xmin);
    eavlFunctorArray<float> ymins(bvh->ymin);
    eavlFunctorArray<float> zmins(bvh->zmin);
    eavlFunctorArray<float> xmaxs(bvh->xmax);
    eavlFunctorArray<float> ymaxs(bvh->ymax);
    eavlFunctorArray<float> zmaxs(bvh->zmax);

    FlatIndxr flatIdx;
    //write out the array in parallel
    eavlExecutor::AddOperation(
        new_eavlMapOp(eavlOpArgs(innerNodes),
                      eavlOpArgs(innerNodes),
                      FloatMemsetFunctor(0)), "");
    eavlExecutor::Go();
    eavlExecutor::AddOperation(
        new_eavlMapOp(eavlOpArgs(bvh->leftChild, bvh->rightChild),
                      eavlOpArgs(eavlIndexable<eavlFloatArray>(innerNodes,  flatIdx.xmin1),
                                 eavlIndexable<eavlFloatArray>(innerNodes,  flatIdx.ymin1),
                                 eavlIndexable<eavlFloatArray>(innerNodes,  flatIdx.zmin1),
                                 eavlIndexable<eavlFloatArray>(innerNodes,  flatIdx.xmax1),
                                 eavlIndexable<eavlFloatArray>(innerNodes,  flatIdx.ymax1),
                                 eavlIndexable<eavlFloatArray>(innerNodes,  flatIdx.zmax1),
                                 eavlIndexable<eavlFloatArray>(innerNodes,  flatIdx.xmin2),
                                 eavlIndexable<eavlFloatArray>(innerNodes,  flatIdx.ymin2),
                                 eavlIndexable<eavlFloatArray>(innerNodes,  flatIdx.zmin2),
                                 eavlIndexable<eavlFloatArray>(innerNodes,  flatIdx.xmax2),
                                 eavlIndexable<eavlFloatArray>(innerNodes,  flatIdx.ymax2),
                                 eavlIndexable<eavlFloatArray>(innerNodes,  flatIdx.zmax2),
                                 eavlIndexable<eavlFloatArray>(innerNodes,  flatIdx.lChild),
                                 eavlIndexable<eavlFloatArray>(innerNodes,  flatIdx.rChild)),
                      InnerToFlatFunctor(xmins, ymins, zmins,
                                         xmaxs, ymaxs, zmaxs, 
                                         numPrimitives), numPrimitives - 1),
                      "write");
    eavlExecutor::Go();

    eavlArrayIndexer numPrim;
    numPrim.mul = 2;
    eavlArrayIndexer id;
    id.mul = 2;
    id.add = 1;
    eavlExecutor::AddOperation(
        new_eavlMapOp(eavlOpArgs(bvh->primId),
                      eavlOpArgs(eavlIndexable<eavlIntArray>(leafNodes,  numPrim),
                                 eavlIndexable<eavlIntArray>(leafNodes,  id)),
                      LeafToFlatFunctor()),
                      "write");
    eavlExecutor::Go();
}

float * MortonBVHBuilder::getInnerNodes(int &_size)
{ 
    if(!convertedToAoS)
    {
        flatten();
        convertedToAoS = true;
    }

    int size = (numPrimitives -1) * 16;
    float * array =  new float[size];
    memcpy((void*)array, innerNodes->GetHostArray(), sizeof(float) * size);
    _size = size;
    return array; 
}
int * MortonBVHBuilder::getLeafNodes(int &_size)
{ 
    if(!convertedToAoS)
    {
        flatten();
        convertedToAoS = true;
    }
    int size = numPrimitives * 2;
    int * array =  new int[size];
    memcpy((void*)array, leafNodes->GetHostArray(), sizeof(int) * size);
    _size = size;
    return array; 
}

eavlFloatArray * MortonBVHBuilder::getInnerNodes()
{
    if(!convertedToAoS)
    {
        flatten();
        convertedToAoS = true;
    }
    wasEavlArrayGiven = true;
    return innerNodes;
}

eavlIntArray * MortonBVHBuilder::getLeafNodes()
{
    if(!convertedToAoS)
    {
        flatten();
        convertedToAoS = true;
    }
    wasEavlArrayGiven = true;
    return leafNodes;
}
