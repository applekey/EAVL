#include "hip/hip_runtime.h"
#include "eavlRayTriangleIntersector.h"
#include "eavlRayDefines.h"
#include "eavlMapOp.h"
#include "eavlTimer.h"
#include "eavlScatterOp.h"
#include "eavlRTUtil.h"

EAVL_HOSTDEVICE int getIntersection(const eavlVector3 rayDir,
				                            const eavlVector3 rayOrigin, 
				                            bool occlusion, 
				                            eavlTextureObject<float4> &innerNodes,
                                    eavlTextureObject<int>  &leafNodes, 
                                    eavlTextureObject<float> &verts,
                                    const float &maxDistance, 
                                    float &distance,
                                    float &minU,
                                    float &minV)
{
    float minDistance = maxDistance;
    int   minIndex    = -1;
    
    float dirx = rayDir.x;
    float diry = rayDir.y;
    float dirz = rayDir.z;

    float invDirx = rcp_safe(dirx);
    float invDiry = rcp_safe(diry);
    float invDirz = rcp_safe(dirz);
    int currentNode;
  
    int todo[64]; //num of nodes to process
    int stackptr = 0;
    int barrier = (int)END_FLAG;
    currentNode = 0;

    todo[stackptr] = barrier;

    float ox = rayOrigin.x;
    float oy = rayOrigin.y;
    float oz = rayOrigin.z;
    float odirx = ox * invDirx;
    float odiry = oy * invDiry;
    float odirz = oz * invDirz;

    while(currentNode != END_FLAG) {

        if(currentNode>-1)
        {
            float4 n1 = innerNodes.getValue(currentNode  ); 
            float4 n2 = innerNodes.getValue(currentNode+1); 
            float4 n3 = innerNodes.getValue(currentNode+2); 
            
            float txmin0 = n1.x * invDirx - odirx;       
            float tymin0 = n1.y * invDiry - odiry;         
            float tzmin0 = n1.z * invDirz - odirz;
            float txmax0 = n1.w * invDirx - odirx;
            float tymax0 = n2.x * invDiry - odiry;
            float tzmax0 = n2.y * invDirz - odirz;
           
            float tmin0 = fmaxf(fmaxf(fmaxf(fminf(tymin0,tymax0),fminf(txmin0,txmax0)),fminf(tzmin0,tzmax0)),0.f);
            float tmax0 = fminf(fminf(fminf(fmaxf(tymin0,tymax0),fmaxf(txmin0,txmax0)),fmaxf(tzmin0,tzmax0)), minDistance);
            
            bool traverseChild0 = (tmax0 >= tmin0);

             
            float txmin1 = n2.z * invDirx - odirx;       
            float tymin1 = n2.w * invDiry - odiry;
            float tzmin1 = n3.x * invDirz - odirz;
            float txmax1 = n3.y * invDirx - odirx;
            float tymax1 = n3.z * invDiry-  odiry;
            float tzmax1 = n3.w * invDirz - odirz;
            float tmin1 = fmaxf(fmaxf(fmaxf(fminf(tymin1,tymax1),fminf(txmin1,txmax1)),fminf(tzmin1,tzmax1)),0.f);
            float tmax1 = fminf(fminf(fminf(fmaxf(tymin1,tymax1),fmaxf(txmin1,txmax1)),fmaxf(tzmin1,tzmax1)), minDistance);
            
            bool traverseChild1 = (tmax1 >= tmin1);

        if(!traverseChild0 && !traverseChild1)
        {

            currentNode = todo[stackptr]; //go back put the stack
            stackptr--;
        }
        else
        {
            float4 n4 = innerNodes.getValue(currentNode+3); 
            int leftChild;
            memcpy(&leftChild, &n4.x,4);
            int rightChild; 
            memcpy(&rightChild, &n4.y, 4);
            currentNode = (traverseChild0) ? leftChild : rightChild;
            if(traverseChild1 && traverseChild0)
            {
                if(tmin0 > tmin1)
                {

                   
                    currentNode = rightChild;
                    stackptr++;
                    todo[stackptr] = leftChild;
                }
                else
                {   
                    stackptr++;
                    todo[stackptr] = rightChild;
                }


            }
        }
        }
        
        if(currentNode < 0 && currentNode != barrier)//check register usage
        {
            currentNode = -currentNode - 1; //swap the neg address 
            int numTri = leafNodes.getValue(currentNode)+1;

            for(int i = 1; i < numTri; i++)
            {        
                    int triIndex = leafNodes.getValue(currentNode+i) * 9;
           
                    eavlVector3 a(verts.getValue(triIndex),verts.getValue(triIndex + 1), verts.getValue(triIndex + 2));
                    eavlVector3 b(verts.getValue(triIndex + 3),verts.getValue(triIndex + 4), verts.getValue(triIndex + 5));
                    eavlVector3 c(verts.getValue(triIndex + 6),verts.getValue(triIndex + 7), verts.getValue(triIndex+8));
                    eavlVector3 e1 = b - a; 
                    eavlVector3 e2=  c - a; 


                    eavlVector3 p;
                    p.x = diry * e2.z - dirz * e2.y;
                    p.y = dirz * e2.x - dirx * e2.z;
                    p.z = dirx * e2.y - diry * e2.x;
                    float dot = e1.x * p.x + e1.y * p.y + e1.z * p.z;
                    if(dot != 0.f)
                    {
                        dot = 1.f/dot;
                        eavlVector3 t;
                        t.x = ox - a.x;
                        t.y = oy - a.y;
                        t.z = oz - a.z;

                        float u = (t.x* p.x + t.y * p.y + t.z * p.z) * dot;
                        if(u >= (0.f - EPSILON) && u <= (1.f + EPSILON))
                        {
                            eavlVector3 q; // = t % e1;
                            q.x = t.y * e1.z - t.z * e1.y;
                            q.y = t.z * e1.x - t.x * e1.z;
                            q.z = t.x * e1.y - t.y * e1.x;
                            float v = (dirx * q.x + diry * q.y + dirz * q.z) * dot;
                            if(v >= (0.f - EPSILON) && v <= (1.f + EPSILON))
                            {
                                float dist = (e2.x * q.x + e2.y * q.y + e2.z * q.z) * dot;
                                if((dist > EPSILON && dist < minDistance) && !(u + v > 1) )
                                {
                                  minDistance = dist;
                                  minIndex = triIndex / 9;
                                  minU = u;
                                  minV = v;
                                  if(occlusion) return minIndex;//or set todo to -1
                                }
                            }
                        }

                    }
                   
            }
            currentNode = todo[stackptr];
            stackptr--;
        }

    }
 distance = minDistance;
 return minIndex;
}

EAVL_HOSTDEVICE int getIntersectionWoop(const eavlVector3 rayDir,
						                            const eavlVector3 rayOrigin, 
						                            bool occlusion, 
						                            eavlTextureObject<float4> &innerNodes,
                                        eavlTextureObject<int>  &leafNodes, 
                                        eavlTextureObject<float4> &verts,
                                        const float &maxDistance, 
                                        float &distance,
                                        float &minU,
                                        float &minV)
    {

    float minDistance = maxDistance;
    int   minIndex    = -1;
    
    float dirx = rayDir.x;
    float diry = rayDir.y;
    float dirz = rayDir.z;

    float invDirx = rcp_safe(dirx);
    float invDiry = rcp_safe(diry);
    float invDirz = rcp_safe(dirz);
    int currentNode;
  
    int todo[64]; //num of nodes to process
    int stackptr = 0;
    int barrier = (int)END_FLAG;
    currentNode = 0;

    todo[stackptr] = barrier;

    float ox = rayOrigin.x;
    float oy = rayOrigin.y;
    float oz = rayOrigin.z;
    float odirx = ox * invDirx;
    float odiry = oy * invDiry;
    float odirz = oz * invDirz;

    while(currentNode != END_FLAG) {

        if(currentNode>-1)
        {
            float4 n1 = innerNodes.getValue(currentNode  ); 
            float4 n2 = innerNodes.getValue(currentNode+1); 
            float4 n3 = innerNodes.getValue(currentNode+2); 
            
            float txmin0 = n1.x * invDirx - odirx;       
            float tymin0 = n1.y * invDiry - odiry;         
            float tzmin0 = n1.z * invDirz - odirz;
            float txmax0 = n1.w * invDirx - odirx;
            float tymax0 = n2.x * invDiry - odiry;
            float tzmax0 = n2.y * invDirz - odirz;
           
            float tmin0 = fmaxf(fmaxf(fmaxf(fminf(tymin0,tymax0),fminf(txmin0,txmax0)),fminf(tzmin0,tzmax0)),0.f);
            float tmax0 = fminf(fminf(fminf(fmaxf(tymin0,tymax0),fmaxf(txmin0,txmax0)),fmaxf(tzmin0,tzmax0)), minDistance);
            
            bool traverseChild0 = (tmax0 >= tmin0);

             
            float txmin1 = n2.z * invDirx - odirx;       
            float tymin1 = n2.w * invDiry - odiry;
            float tzmin1 = n3.x * invDirz - odirz;
            float txmax1 = n3.y * invDirx - odirx;
            float tymax1 = n3.z * invDiry-  odiry;
            float tzmax1 = n3.w * invDirz - odirz;
            float tmin1 = fmaxf(fmaxf(fmaxf(fminf(tymin1,tymax1),fminf(txmin1,txmax1)),fminf(tzmin1,tzmax1)),0.f);
            float tmax1 = fminf(fminf(fminf(fmaxf(tymin1,tymax1),fmaxf(txmin1,txmax1)),fmaxf(tzmin1,tzmax1)), minDistance);
            
            bool traverseChild1 = (tmax1 >= tmin1);

        if(!traverseChild0 && !traverseChild1)
        {

            currentNode = todo[stackptr]; //go back put the stack
            stackptr--;
        }
        else
        {
            float4 n4 = innerNodes.getValue(currentNode+3); 
            int leftChild;
            memcpy(&leftChild, &n4.x,4);
            int rightChild; 
            memcpy(&rightChild, &n4.y, 4);
            currentNode = (traverseChild0) ? leftChild : rightChild;
            if(traverseChild1 && traverseChild0)
            {
                if(tmin0 > tmin1)
                {

                   
                    currentNode = rightChild;
                    stackptr++;
                    todo[stackptr] = leftChild;
                }
                else
                {   
                    stackptr++;
                    todo[stackptr] = rightChild;
                }


            }
        }
        }
        
        if(currentNode < 0 && currentNode != barrier)//check register usage
        {
            currentNode = -currentNode - 1; //swap the neg address 
            int numTri = leafNodes.getValue(currentNode)+1;

            for(int i = 1; i < numTri; i++)
            {        
                    int triIndex = leafNodes.getValue(currentNode+i) * 3;
 
                    float4 zCol = verts.getValue(triIndex);

                    float unitOriginZ = -ox * zCol.x  - oy * zCol.y - oz * zCol.z + zCol.w ;
                    float unitDirZ = dirx * zCol.x + diry * zCol.y + dirz * zCol.z;
                    //printf(" %f %f %f %f ",dirx, diry, dirz, zCol.w);
                    unitDirZ = 1.f / unitDirZ;

                    float dist = unitOriginZ * unitDirZ;
                    //printf("Distance %f ", dist);
                    if((dist > EPSILON && dist < minDistance))
                    {   
                        float4 yCol = verts.getValue(triIndex+1);
                        float unitOriginY = ox * yCol.x  + oy * yCol.y + oz * yCol.z + yCol.w ;
                        float unitDirY = dirx * yCol.x + diry * yCol.y + dirz * yCol.z;
                        float v = dist * unitDirY + unitOriginY;

                        if(v >= 0.f)
                        {
                            float4 xCol = verts.getValue(triIndex+2);
                            float unitOriginX = ox * xCol.x  + oy * xCol.y + oz * xCol.z + xCol.w ;
                            float unitDirX = dirx * xCol.x + diry * xCol.y + dirz * xCol.z;
                            float u = dist * unitDirX + unitOriginX;

                            if((u >= 0.f) && ((u+v) <= 1.00002))
                            {
                                minDistance = dist;
                                minIndex = triIndex / 3;
                                minU = u;
                                minV = v;
                                if(occlusion) return minIndex;//or set todo to -1
                            }
                        }

                    }    
                   
            }
            currentNode = todo[stackptr];
            stackptr--;
        }

    }
 distance = minDistance;
 return minIndex;
}

EAVL_HOSTDEVICE int getIntersectionOcculsionWoop(const eavlVector3 rayDir,
								   			                         const eavlVector3 rayOrigin, 
								   			                         eavlTextureObject<float4> &innerNodes,
                                   			         eavlTextureObject<int>  &leafNodes, 
                                   			         eavlTextureObject<float4> &verts,
                                   			         const float &maxDistance)
{

    float dirx = rayDir.x;
    float diry = rayDir.y;
    float dirz = rayDir.z;

    float invDirx = rcp_safe(dirx);
    float invDiry = rcp_safe(diry);
    float invDirz = rcp_safe(dirz);
    int currentNode;
  
    int todo[64]; //num of nodes to process
    int stackptr = 0;
    int barrier = (int)END_FLAG;
    currentNode = 0;

    todo[stackptr] = barrier;

    float ox = rayOrigin.x;
    float oy = rayOrigin.y;
    float oz = rayOrigin.z;
    float odirx = ox * invDirx;
    float odiry = oy * invDiry;
    float odirz = oz * invDirz;

    while(currentNode != END_FLAG) {

        if(currentNode>-1)
        {

            float4 n1 = innerNodes.getValue(currentNode  ); //(txmin0, tymin0, tzmin0, txmax0)
            float4 n2 = innerNodes.getValue(currentNode+1); //(tymax0, tzmax0, txmin1, tymin1)
            float4 n3 = innerNodes.getValue(currentNode+2); //(tzmin1, txmax1, tymax1, tzmax1)
            
            float txmin0 = n1.x * invDirx - odirx;       
            float tymin0 = n1.y * invDiry - odiry;         
            float tzmin0 = n1.z * invDirz - odirz;
            float txmax0 = n1.w * invDirx - odirx;
            float tymax0 = n2.x * invDiry - odiry;
            float tzmax0 = n2.y * invDirz - odirz;
           
            float tmin0 = max(max(max(min(tymin0,tymax0),min(txmin0,txmax0)),min(tzmin0,tzmax0)),0.f);
            float tmax0 = min(min(min(max(tymin0,tymax0),max(txmin0,txmax0)),max(tzmin0,tzmax0)), maxDistance);
            
            bool traverseChild0 = (tmax0 >= tmin0);

             
            float txmin1 = n2.z * invDirx - odirx;       
            float tymin1 = n2.w * invDiry - odiry;
            float tzmin1 = n3.x * invDirz - odirz;
            float txmax1 = n3.y * invDirx - odirx;
            float tymax1 = n3.z * invDiry-  odiry;
            float tzmax1 = n3.w * invDirz - odirz;
            float tmin1 = max(max(max(min(tymin1,tymax1),min(txmin1,txmax1)),min(tzmin1,tzmax1)),0.f);
            float tmax1 = min(min(min(max(tymin1,tymax1),max(txmin1,txmax1)),max(tzmin1,tzmax1)), maxDistance);
            
            bool traverseChild1 = (tmax1 >= tmin1);

        if(!traverseChild0 && !traverseChild1)
        {

            currentNode = todo[stackptr]; //go back put the stack
            stackptr--;
        }
        else
        {
            float4 n4 = innerNodes.getValue(currentNode+3); 
            int leftChild;
            memcpy(&leftChild, &n4.x,4);
            int rightChild; 
            memcpy(&rightChild, &n4.y, 4);
            currentNode = (traverseChild0) ? leftChild : rightChild;
            if(traverseChild1 && traverseChild0)
            {
                if(tmin0 > tmin1)
                {

                   
                    currentNode = rightChild;
                    stackptr++;
                    todo[stackptr] = leftChild;
                }
                else
                {   
                    stackptr++;
                    todo[stackptr] = rightChild;
                }
            }
        }
        }
        
        if(currentNode < 0 && currentNode != barrier)//check register usage
        {
            currentNode = -currentNode - 1; //swap the neg address 
            int numTri = leafNodes.getValue(currentNode)+1;

            for(int i = 1; i < numTri; i++)
            {        
                    int triIndex = leafNodes.getValue(currentNode+i) * 3;
                    float4 zCol = verts.getValue(triIndex);
                    float unitOriginZ = -ox * zCol.x  - oy * zCol.y - oz * zCol.z + zCol.w ;
                    float unitDirZ = dirx * zCol.x + diry * zCol.y + dirz * zCol.z;
                    unitDirZ = 1.f / unitDirZ;

                    float dist = unitOriginZ * unitDirZ;
                    if((dist > EPSILON && dist < maxDistance))
                    {   
                        float4 yCol = verts.getValue(triIndex+1);
                        float unitOriginY = ox * yCol.x  + oy * yCol.y + oz * yCol.z + yCol.w ;
                        float unitDirY = dirx * yCol.x + diry * yCol.y + dirz * yCol.z;
                        float v = dist * unitDirY + unitOriginY;

                        if(v >= 0.f)
                        {
                            float4 xCol = verts.getValue(triIndex+2);
                            float unitOriginX = ox * xCol.x  + oy * xCol.y + oz * xCol.z + xCol.w ;
                            float unitDirX = dirx * xCol.x + diry * xCol.y + dirz * xCol.z;
                            float u = dist * unitDirX + unitOriginX;

                            if((u >= 0.f) && ((u+v) <= 1.00002))
                            {
                                return 0; //ray is occluded
                            }
                        }

                    }    
                   
            }
            currentNode = todo[stackptr];
            stackptr--;
        }

    }
 
 return 1; //clear path
}

EAVL_HOSTDEVICE int getIntersectionOcclusion(const eavlVector3 rayDir,
								   			                         const eavlVector3 rayOrigin, 
								   			                         eavlTextureObject<float4> &innerNodes,
                                   			         eavlTextureObject<int>  &leafNodes, 
                                   			         eavlTextureObject<float> &verts,
                                   			         const float &maxDistance)
{
    float minDistance = maxDistance;
    
    float dirx = rayDir.x;
    float diry = rayDir.y;
    float dirz = rayDir.z;

    float invDirx = rcp_safe(dirx);
    float invDiry = rcp_safe(diry);
    float invDirz = rcp_safe(dirz);
    int currentNode;
  
    int todo[64]; //num of nodes to process
    int stackptr = 0;
    int barrier = (int)END_FLAG;
    currentNode = 0;

    todo[stackptr] = barrier;

    float ox = rayOrigin.x;
    float oy = rayOrigin.y;
    float oz = rayOrigin.z;
    float odirx = ox * invDirx;
    float odiry = oy * invDiry;
    float odirz = oz * invDirz;

    while(currentNode != END_FLAG) {

        if(currentNode>-1)
        {
            float4 n1 = innerNodes.getValue(currentNode  ); 
            float4 n2 = innerNodes.getValue(currentNode+1); 
            float4 n3 = innerNodes.getValue(currentNode+2); 
            
            float txmin0 = n1.x * invDirx - odirx;       
            float tymin0 = n1.y * invDiry - odiry;         
            float tzmin0 = n1.z * invDirz - odirz;
            float txmax0 = n1.w * invDirx - odirx;
            float tymax0 = n2.x * invDiry - odiry;
            float tzmax0 = n2.y * invDirz - odirz;
           
            float tmin0 = fmaxf(fmaxf(fmaxf(fminf(tymin0,tymax0),fminf(txmin0,txmax0)),fminf(tzmin0,tzmax0)),0.f);
            float tmax0 = fminf(fminf(fminf(fmaxf(tymin0,tymax0),fmaxf(txmin0,txmax0)),fmaxf(tzmin0,tzmax0)), minDistance);
            
            bool traverseChild0 = (tmax0 >= tmin0);

             
            float txmin1 = n2.z * invDirx - odirx;       
            float tymin1 = n2.w * invDiry - odiry;
            float tzmin1 = n3.x * invDirz - odirz;
            float txmax1 = n3.y * invDirx - odirx;
            float tymax1 = n3.z * invDiry-  odiry;
            float tzmax1 = n3.w * invDirz - odirz;
            float tmin1 = fmaxf(fmaxf(fmaxf(fminf(tymin1,tymax1),fminf(txmin1,txmax1)),fminf(tzmin1,tzmax1)),0.f);
            float tmax1 = fminf(fminf(fminf(fmaxf(tymin1,tymax1),fmaxf(txmin1,txmax1)),fmaxf(tzmin1,tzmax1)), minDistance);
            
            bool traverseChild1 = (tmax1 >= tmin1);

        if(!traverseChild0 && !traverseChild1)
        {

            currentNode = todo[stackptr]; //go back put the stack
            stackptr--;
        }
        else
        {
            float4 n4 = innerNodes.getValue(currentNode+3); 
            int leftChild;
            memcpy(&leftChild, &n4.x,4);
            int rightChild; 
            memcpy(&rightChild, &n4.y, 4);
            currentNode = (traverseChild0) ? leftChild : rightChild;
            if(traverseChild1 && traverseChild0)
            {
                if(tmin0 > tmin1)
                {

                   
                    currentNode = rightChild;
                    stackptr++;
                    todo[stackptr] = leftChild;
                }
                else
                {   
                    stackptr++;
                    todo[stackptr] = rightChild;
                }


            }
        }
        }
        
        if(currentNode < 0 && currentNode != barrier)//check register usage
        {
            currentNode = -currentNode - 1; //swap the neg address 
            int numTri = leafNodes.getValue(currentNode)+1;

            for(int i = 1; i < numTri; i++)
            {        
                    int triIndex = leafNodes.getValue(currentNode+i) * 9;
           
                    eavlVector3 a(verts.getValue(triIndex + 0),verts.getValue(triIndex + 1), verts.getValue(triIndex + 2));
                    eavlVector3 b(verts.getValue(triIndex + 3),verts.getValue(triIndex + 4), verts.getValue(triIndex + 5));
                    eavlVector3 c(verts.getValue(triIndex + 6),verts.getValue(triIndex + 7), verts.getValue(triIndex + 8));
                    eavlVector3 e1 = b - a; 
                    eavlVector3 e2=  c - a; 


                    eavlVector3 p;
                    p.x = diry * e2.z - dirz * e2.y;
                    p.y = dirz * e2.x - dirx * e2.z;
                    p.z = dirx * e2.y - diry * e2.x;
                    float dot = e1.x * p.x + e1.y * p.y + e1.z * p.z;
                    if(dot != 0.f)
                    {
                        dot = 1.f/dot;
                        eavlVector3 t;
                        t.x = ox - a.x;
                        t.y = oy - a.y;
                        t.z = oz - a.z;

                        float u = (t.x* p.x + t.y * p.y + t.z * p.z) * dot;
                        if(u >= (0.f - EPSILON) && u <= (1.f + EPSILON))
                        {
                            eavlVector3 q; // = t % e1;
                            q.x = t.y * e1.z - t.z * e1.y;
                            q.y = t.z * e1.x - t.x * e1.z;
                            q.z = t.x * e1.y - t.y * e1.x;
                            float v = (dirx * q.x + diry * q.y + dirz * q.z) * dot;
                            if(v >= (0.f - EPSILON) && v <= (1.f + EPSILON))
                            {
                                float dist = (e2.x * q.x + e2.y * q.y + e2.z * q.z) * dot;
                                if((dist > EPSILON && dist < minDistance) && !(u + v > 1) )
                                {
                                  return 0; //ray is occluded
                                }
                            }
                        }

                    }
                   
            }
            currentNode = todo[stackptr];
            stackptr--;
        }

    }
 
 return 1;
}

struct MultipleDistancesTriangleDepthFunctorWoop{


    eavlTextureObject<float4> verts;
    eavlTextureObject<float4> innerNodes;
    eavlTextureObject<int>  leafNodes;

    MultipleDistancesTriangleDepthFunctorWoop(eavlTextureObject<float4> *_verts,
    						 				  eavlTextureObject<float4> *_innerNodes,
                                              eavlTextureObject<int>  *_leafNodes)
        :verts(*_verts),
         innerNodes(*_innerNodes),
         leafNodes(*_leafNodes)
    {}                                                 
    EAVL_HOSTDEVICE tuple<int,float, float, float> operator()( tuple<float,float,float,float,float,float,float,int> rayTuple){
       
        int hitIdx = get<7>(rayTuple);
        if(hitIdx < 0) return tuple<int,float, float, float>(hitIdx, INFINITE, 0.0f, 0.0f);
        float distance;
        float maxDistance = get<6>(rayTuple);
        eavlVector3 rayOrigin(get<0>(rayTuple),get<1>(rayTuple),get<2>(rayTuple));
        eavlVector3       ray(get<3>(rayTuple),get<4>(rayTuple),get<5>(rayTuple));
        float u = 0.f;
        float v = 0.f;
        int minHit = getIntersectionWoop(ray,
    							 	 rayOrigin, 
    							 	 false,
    							 	 innerNodes,
    							 	 leafNodes, 
    							 	 verts,
    							 	 maxDistance,
    							 	 distance,
                                     u,
                                     v);
        
		return tuple<int,float, float, float>(minHit, distance, u, v);
    }
};

struct MultipleDistancesTriangleDepthFunctor{


    eavlTextureObject<float> verts;
    eavlTextureObject<float4> innerNodes;
    eavlTextureObject<int>  leafNodes;

    MultipleDistancesTriangleDepthFunctor(eavlTextureObject<float> *_verts,
    						 				  eavlTextureObject<float4> *_innerNodes,
                                              eavlTextureObject<int>  *_leafNodes)
        :verts(*_verts),
         innerNodes(*_innerNodes),
         leafNodes(*_leafNodes)
    {}                                                 
    EAVL_HOSTDEVICE tuple<int,float, float, float> operator()( tuple<float,float,float,float,float,float,float,int> rayTuple){
       
        int hitIdx = get<7>(rayTuple);
        if(hitIdx < 0) return tuple<int,float, float, float>(hitIdx, INFINITE, 0.0f, 0.0f);
        float distance;
        float maxDistance = get<6>(rayTuple);
        eavlVector3 rayOrigin(get<0>(rayTuple),get<1>(rayTuple),get<2>(rayTuple));
        eavlVector3       ray(get<3>(rayTuple),get<4>(rayTuple),get<5>(rayTuple));
        float u = 0.f;
        float v = 0.f;
        int minHit = getIntersection(ray,
    							 	 rayOrigin, 
    							 	 false,
    							 	 innerNodes,
    							 	 leafNodes, 
    							 	 verts,
    							 	 maxDistance,
    							 	 distance,
                                     u,
                                     v);
        
		return tuple<int,float, float, float>(minHit, distance, u, v);
    }
};

struct SingleDistanceTriangleDepthFunctorWoop{


    eavlTextureObject<float4> verts;
    eavlTextureObject<float4> innerNodes;
    eavlTextureObject<int>  leafNodes;
    int maxDistance;

    SingleDistanceTriangleDepthFunctorWoop(eavlTextureObject<float4> *_verts,
    						 				  eavlTextureObject<float4> *_innerNodes,
                                              eavlTextureObject<int>  *_leafNodes,
                                              int _maxDistance)
        :verts(*_verts),
         innerNodes(*_innerNodes),
         leafNodes(*_leafNodes),
         maxDistance(_maxDistance)

 
    {}                                                 
    EAVL_HOSTDEVICE tuple<int,float,float,float> operator()( tuple<float,float,float,float,float,float,int> rayTuple){
       
        int hitIdx = get<6>(rayTuple);
        if(hitIdx < 0) return tuple<int,float,float,float>(hitIdx,INFINITE,0.0f,0.0f);
        float distance;
        eavlVector3 rayOrigin(get<0>(rayTuple),get<1>(rayTuple),get<2>(rayTuple));
        eavlVector3       ray(get<3>(rayTuple),get<4>(rayTuple),get<5>(rayTuple));
        //printf(" %f %f %f " ,ray.x,ray.y,ray.z);
        float u = 0.f;
        float v = 0.f;
        int minHit = getIntersectionWoop(ray,
    							 	 rayOrigin, 
    							 	 false,
    							 	 innerNodes,
    							 	 leafNodes, 
    							 	 verts,
    							 	 maxDistance,
    							 	 distance,
                                     u,
                                     v);
        if(minHit == -1) distance = INFINITE;
		return tuple<int,float,float,float>(minHit, distance,u,v);
 
    }
};

struct SingleDistanceTriangleDepthFunctor{


    eavlTextureObject<float> verts;
    eavlTextureObject<float4> innerNodes;
    eavlTextureObject<int>  leafNodes;
    int maxDistance;

    SingleDistanceTriangleDepthFunctor(eavlTextureObject<float> *_verts,
    						 				  eavlTextureObject<float4> *_innerNodes,
                                              eavlTextureObject<int>  *_leafNodes,
                                              int _maxDistance)
        :verts(*_verts),
         innerNodes(*_innerNodes),
         leafNodes(*_leafNodes),
         maxDistance(_maxDistance)

 
    {}                                                 
    EAVL_HOSTDEVICE tuple<int,float,float,float> operator()( tuple<float,float,float,float,float,float,int> rayTuple){
       
        int hitIdx = get<6>(rayTuple);
        if(hitIdx < 0) return tuple<int,float,float,float>(hitIdx,INFINITE,0.0f,0.0f);
        float distance;
        eavlVector3 rayOrigin(get<0>(rayTuple),get<1>(rayTuple),get<2>(rayTuple));
        eavlVector3       ray(get<3>(rayTuple),get<4>(rayTuple),get<5>(rayTuple));
        //printf(" %f %f %f " ,ray.x,ray.y,ray.z);
        float u = 0.f;
        float v = 0.f;
        int minHit = getIntersection(ray,
    							 	 rayOrigin, 
    							 	 false,
    							 	 innerNodes,
    							 	 leafNodes, 
    							 	 verts,
    							 	 maxDistance,
    							 	 distance,
                                     u,
                                     v);
        if(minHit == -1) distance = INFINITE;
		return tuple<int,float,float,float>(minHit, distance,u,v);
 
    }
};

struct ShadowFunctorWoop{
    eavlTextureObject<float4> verts;
    eavlTextureObject<float4> innerNodes;
    eavlTextureObject<int>  leafNodes;
    eavlVector3				lightPosition;

    ShadowFunctorWoop(eavlTextureObject<float4> *_verts,
    				  eavlTextureObject<float4> *_innerNodes,
                      eavlTextureObject<int>  *_leafNodes,
                      eavlVector3 &_lightPosition)
        :verts(*_verts),
         innerNodes(*_innerNodes),
         leafNodes(*_leafNodes),
         lightPosition(_lightPosition)

 
    {}                                                 
    EAVL_HOSTDEVICE tuple<int> operator()( tuple<float,float,float,int> rayTuple){
       
        int hitIdx = get<3>(rayTuple);
        if(hitIdx < 0) return tuple<int>(0);
        eavlVector3 rayOrigin(get<0>(rayTuple),get<1>(rayTuple),get<2>(rayTuple));
        eavlVector3 rayDir = lightPosition - rayOrigin;
        float maxDistance = sqrt(rayDir*rayDir);
        int minHit = getIntersectionOcculsionWoop(rayDir,
                            							 	 		  rayOrigin,
                            							 	 		  innerNodes,
                            							 	 		  leafNodes, 
                            							 	 		  verts,
                            							 	 		  maxDistance);
		return tuple<int>(minHit);
 
    }
};

struct ShadowFunctor{
    eavlTextureObject<float> verts;
    eavlTextureObject<float4> innerNodes;
    eavlTextureObject<int>  leafNodes;
    eavlVector3				lightPosition;

    ShadowFunctor(eavlTextureObject<float> *_verts,
    				  eavlTextureObject<float4> *_innerNodes,
                      eavlTextureObject<int>  *_leafNodes,
                      eavlVector3 &_lightPosition)
        :verts(*_verts),
         innerNodes(*_innerNodes),
         leafNodes(*_leafNodes),
         lightPosition(_lightPosition)

 
    {}                                                 
    EAVL_HOSTDEVICE tuple<int> operator()( tuple<float,float,float,int> rayTuple){
       
        int hitIdx = get<3>(rayTuple);
        if(hitIdx < 0) return tuple<int>(0);
        eavlVector3 rayOrigin(get<0>(rayTuple),get<1>(rayTuple),get<2>(rayTuple));
        eavlVector3 rayDir = lightPosition - rayOrigin;
        float maxDistance = sqrt(rayDir*rayDir);
        int minHit = getIntersectionOcclusion(rayDir,
                        							 	 		  rayOrigin,
                        							 	 		  innerNodes,
                        							 	 		  leafNodes, 
                        							 	 		  verts,
                        							 	 		  maxDistance);
		return tuple<int>(minHit);
 
    }
};

struct OcclusionFunctorWoop{
    eavlTextureObject<float4>   verts;
    eavlTextureObject<float4>   innerNodes;
    eavlTextureObject<int>      leafNodes;
    float                       maxDistance;

    OcclusionFunctorWoop(eavlTextureObject<float4> *_verts,
                         eavlTextureObject<float4> *_innerNodes,
                         eavlTextureObject<int>    *_leafNodes,
                         float &_maxDistance)
        :verts(*_verts),
         innerNodes(*_innerNodes),
         leafNodes(*_leafNodes),
         maxDistance(_maxDistance)
 
    {}                                                 
    EAVL_HOSTDEVICE tuple<int> operator()( tuple<float,float,float,float,float,float,int> rayTuple){
       
        int hitIdx = get<6>(rayTuple);
        if(hitIdx < 0) return tuple<int>(0);
        eavlVector3 rayOrigin(get<0>(rayTuple),get<1>(rayTuple),get<2>(rayTuple));
        eavlVector3 rayDir(get<3>(rayTuple),get<4>(rayTuple),get<5>(rayTuple));
        int minHit = getIntersectionOcculsionWoop(rayDir,
                                              rayOrigin,
                                              innerNodes,
                                              leafNodes, 
                                              verts,
                                              maxDistance);
        return tuple<int>(minHit);
 
    }
};

struct OcclusionFunctor{
    eavlTextureObject<float>   verts;
    eavlTextureObject<float4>   innerNodes;
    eavlTextureObject<int>      leafNodes;
    float                       maxDistance;

    OcclusionFunctor(eavlTextureObject<float> *_verts,
                         eavlTextureObject<float4> *_innerNodes,
                         eavlTextureObject<int>    *_leafNodes,
                         float &_maxDistance)
        :verts(*_verts),
         innerNodes(*_innerNodes),
         leafNodes(*_leafNodes),
         maxDistance(_maxDistance)
 
    {}                                                 
    EAVL_HOSTDEVICE tuple<int> operator()( tuple<float,float,float,float,float,float,int> rayTuple){
       
        int hitIdx = get<6>(rayTuple);
        if(hitIdx < 0) return tuple<int>(0);
        eavlVector3 rayOrigin(get<0>(rayTuple),get<1>(rayTuple),get<2>(rayTuple));
        eavlVector3 rayDir(get<3>(rayTuple),get<4>(rayTuple),get<5>(rayTuple));
        int minHit = getIntersectionOcclusion(rayDir,
                                              rayOrigin,
                                              innerNodes,
                                              leafNodes, 
                                              verts,
                                              maxDistance);
        return tuple<int>(minHit);
 
    }
};

EAVL_HOSTDEVICE float testFunction(const eavlTextureObject<float4> *tt)
{ 
    return tt->getValue(0).x;
}

struct testfunctor{


    eavlTextureObject<float4> verts;
   

    testfunctor(eavlTextureObject<float4> *_verts)
        :verts(*_verts)

 
    {}                                                 
    EAVL_HOSTDEVICE tuple<float> operator()( tuple<float> rayTuple){
       
        float distance = get<0>(rayTuple);
        printf("ID inside %llu \n", verts.textureObjectId);
        distance *= verts.getValue(0).x;

          
        return tuple<float>( distance);
 
    }
};


EAVL_HOSTONLY void eavlRayTriangleIntersector::intersectionDepth(const eavlRay *rays, 
					   								   	  		 const int &maxDistance, 
											  		      		 const eavlRayTriangleGeometry *geometry)
{
  if(geometry->woopify)
  {
	  eavlExecutor::AddOperation(new_eavlMapOp(eavlOpArgs(rays->rayOriginX,
                                            rays->rayOriginY,
                                            rays->rayOriginZ,
                                            rays->rayDirX,
                                            rays->rayDirY,
                                            rays->rayDirZ,
                                            rays->hitIdx),
                                            eavlOpArgs(rays->hitIdx,
                                            rays->distance,
                                            rays->alpha,
                                            rays->beta),
                                            SingleDistanceTriangleDepthFunctorWoop(geometry->vertices,
                                            geometry->bvhInnerNodes,
                                            geometry->bvhLeafNodes,
                                            maxDistance)),
                                            "Intersect");
      eavlExecutor::Go();
   }
   else
   {
      eavlExecutor::AddOperation(new_eavlMapOp(eavlOpArgs(rays->rayOriginX,
                                            rays->rayOriginY,
                                            rays->rayOriginZ,
                                            rays->rayDirX,
                                            rays->rayDirY,
                                            rays->rayDirZ,
                                            rays->hitIdx),
                                            eavlOpArgs(rays->hitIdx,
                                            rays->distance,
                                            rays->alpha,
                                            rays->beta),
                                            SingleDistanceTriangleDepthFunctor(geometry->verticesActual,
                                            geometry->bvhInnerNodes,
                                            geometry->bvhLeafNodes,
                                            maxDistance)),
                                            "Intersect");
      eavlExecutor::Go();
   }
}

EAVL_HOSTONLY void eavlRayTriangleIntersector::intersectionDepth(const eavlRay *rays, 
													      		 eavlFloatArray *maxDistances,
													      		 const eavlRayTriangleGeometry *geometry)
{
  if(geometry->woopify)
  {
	  eavlExecutor::AddOperation(new_eavlMapOp(eavlOpArgs(rays->rayOriginX,
                                                        rays->rayOriginY,
                                                        rays->rayOriginZ,
                                                        rays->rayDirX,
                                                        rays->rayDirY,
                                                        rays->rayDirZ,
                                                        maxDistances,
                                                        rays->hitIdx),
                                                        eavlOpArgs(rays->hitIdx,
                                                        rays->distance,
                                                        rays->alpha,
                                                        rays->beta),
                                                        MultipleDistancesTriangleDepthFunctorWoop(geometry->vertices,
                                                        geometry->bvhInnerNodes,
                                                        geometry->bvhLeafNodes)),
                                                        "Intersect");
      eavlExecutor::Go();
   }
   else
   {
      eavlExecutor::AddOperation(new_eavlMapOp(eavlOpArgs(rays->rayOriginX,
                                                        rays->rayOriginY,
                                                        rays->rayOriginZ,
                                                        rays->rayDirX,
                                                        rays->rayDirY,
                                                        rays->rayDirZ,
                                                        maxDistances,
                                                        rays->hitIdx),
                                                        eavlOpArgs(rays->hitIdx,
                                                        rays->distance,
                                                        rays->alpha,
                                                        rays->beta),
                                                        MultipleDistancesTriangleDepthFunctor(geometry->verticesActual,
                                                        geometry->bvhInnerNodes,
                                                        geometry->bvhLeafNodes)),
                                                        "Intersect");
      eavlExecutor::Go();
   }
}

EAVL_HOSTONLY void eavlRayTriangleIntersector::intersectionShadow(const eavlFullRay *rays, 
													      		 eavlIntArray *hits,
													      		 eavlVector3 &lightPosition,
													      		 const eavlRayTriangleGeometry *geometry)
{
  if(geometry->woopify)
  {
	  eavlExecutor::AddOperation(new_eavlMapOp(eavlOpArgs(rays->intersectionX,
                                                        rays->intersectionY,
                                                        rays->intersectionZ,
                                                        rays->hitIdx),
                                                        eavlOpArgs(hits),
                                                        ShadowFunctorWoop(geometry->vertices,
                                                        geometry->bvhInnerNodes,
                                                        geometry->bvhLeafNodes,
                                                        lightPosition)),
                                                        "Intersect");
     eavlExecutor::Go();
  }
  else
  {
    eavlExecutor::AddOperation(new_eavlMapOp(eavlOpArgs(rays->intersectionX,
                                                        rays->intersectionY,
                                                        rays->intersectionZ,
                                                        rays->hitIdx),
                                                        eavlOpArgs(hits),
                                                        ShadowFunctor(geometry->verticesActual,
                                                        geometry->bvhInnerNodes,
                                                        geometry->bvhLeafNodes,
                                                        lightPosition)),
                                                        "Intersect");
     eavlExecutor::Go();
  }
}

EAVL_HOSTONLY void eavlRayTriangleIntersector::intersectionOcclusion(const eavlFullRay *rays, 
                                                                     eavlFloatArray *occX,
                                                                     eavlFloatArray *occY,
                                                                     eavlFloatArray *occZ,
                                                                     eavlIntArray *hits,
                                                                     eavlArrayIndexer *occIndexer,
                                                                     float maxDistance,  
                                                                     const eavlRayTriangleGeometry *geometry)
{
  if(geometry->woopify)
  {
    eavlExecutor::AddOperation(new_eavlMapOp(eavlOpArgs(eavlIndexable<eavlFloatArray>(rays->intersectionX, *occIndexer),
                                                        eavlIndexable<eavlFloatArray>(rays->intersectionY, *occIndexer),
                                                        eavlIndexable<eavlFloatArray>(rays->intersectionZ, *occIndexer),
                                                        eavlIndexable<eavlFloatArray>(occX),
                                                        eavlIndexable<eavlFloatArray>(occY),
                                                        eavlIndexable<eavlFloatArray>(occZ),
                                                        eavlIndexable<eavlIntArray>(rays->hitIdx, *occIndexer)),
                                                        eavlOpArgs(hits),
                                                        OcclusionFunctorWoop(geometry->vertices,
                                                                             geometry->bvhInnerNodes,
                                                                             geometry->bvhLeafNodes,
                                                                             maxDistance)),
                                                        "Intersect");
    eavlExecutor::Go();
  }
  else
  {
    eavlExecutor::AddOperation(new_eavlMapOp(eavlOpArgs(eavlIndexable<eavlFloatArray>(rays->intersectionX, *occIndexer),
                                                        eavlIndexable<eavlFloatArray>(rays->intersectionY, *occIndexer),
                                                        eavlIndexable<eavlFloatArray>(rays->intersectionZ, *occIndexer),
                                                        eavlIndexable<eavlFloatArray>(occX),
                                                        eavlIndexable<eavlFloatArray>(occY),
                                                        eavlIndexable<eavlFloatArray>(occZ),
                                                        eavlIndexable<eavlIntArray>(rays->hitIdx, *occIndexer)),
                                                        eavlOpArgs(hits),
                                                        OcclusionFunctor(geometry->verticesActual,
                                                                         geometry->bvhInnerNodes,
                                                                         geometry->bvhLeafNodes,
                                                                         maxDistance)),
                                                        "Intersect");
    eavlExecutor::Go();
  }
}

EAVL_HOSTONLY void eavlRayTriangleIntersector::testIntersections(const eavlRay *rays, 
                                                                const int &maxDistance, 
                                                                const eavlRayTriangleGeometry *geometry,
                                                                const int &warmUpRounds,
                                                                const int &testRounds,
                                                                eavlRayCamera *cam)
{
	int height = cam->getHeight();
	int width = cam->getWidth(); 
	int size = width * height;

	  eavlIntArray    *dummy= new eavlIntArray("",1,size);
    eavlFloatArray  *dummyFloat= new eavlFloatArray("",1,size);
    eavlExecutor::AddOperation(new_eavlMapOp(eavlOpArgs(rays->rayOriginX),
                                                 eavlOpArgs(
                                                            dummyFloat),
                                                testfunctor(geometry->vertices),1),
                                                "TestFunc");
    eavlExecutor::Go();
    eavlExecutor::AddOperation(new_eavlMapOp(eavlOpArgs(rays->hitIdx), //dummy arg
                                    eavlOpArgs(rays->hitIdx),
                                    IntMemsetFunctor(0.f)), 
                                    "resetHits");
    eavlExecutor::Go();


    cout<<"Warming up "<<warmUpRounds<<" rounds."<<endl;
    int warm = eavlTimer::Start(); 

    for(int i = 0; i < warmUpRounds; i++)
    {
    	eavlExecutor::AddOperation(new_eavlMapOp(eavlOpArgs(rays->rayOriginX,
															rays->rayOriginY,
															rays->rayOriginZ,
															rays->rayDirX,
															rays->rayDirY,
															rays->rayDirZ,
                                                            rays->hitIdx),
                                            	 eavlOpArgs(dummy,
                                             				dummyFloat,
                                                            rays->alpha,
                                                            rays->beta),
                                             	SingleDistanceTriangleDepthFunctorWoop(geometry->vertices,
                                             										   geometry->bvhInnerNodes,
                                             										   geometry->bvhLeafNodes,
                                             										   INFINITE)),
                                             	"Intersect");
    	eavlExecutor::Go();
    }

    float rayper=size/(eavlTimer::Stop(warm,"warm")/(float)warmUpRounds);
    cout << "Warm up "<<rayper/1000000.f<< " Mrays/sec"<<endl;

    int test = eavlTimer::Start();

    for(int i = 0; i < testRounds; i++)
    {
    	eavlExecutor::AddOperation(new_eavlMapOp(eavlOpArgs(rays->rayOriginX,
															rays->rayOriginY,
															rays->rayOriginZ,
															rays->rayDirX,
															rays->rayDirY,
															rays->rayDirZ,
                                                            rays->hitIdx),
                                            	 eavlOpArgs(dummy,
                                             				dummyFloat,
                                                            rays->alpha,
                                                            rays->beta),
                                             	SingleDistanceTriangleDepthFunctorWoop(geometry->vertices,
                                             										   geometry->bvhInnerNodes,
                                             										   geometry->bvhLeafNodes,
                                             										   INFINITE)),
                                             	"Intersect");
    	eavlExecutor::Go();
    }

    rayper=size/(eavlTimer::Stop(test,"test")/(float)testRounds);
    cout << "# "<<rayper/1000000.f<<endl;

    eavlExecutor::AddOperation(new_eavlMapOp(eavlOpArgs(rays->rayOriginX,
														rays->rayOriginY,
														rays->rayOriginZ,
														rays->rayDirX,
														rays->rayDirY,
														rays->rayDirZ,
                                                        rays->hitIdx),
                                             eavlOpArgs(rays->hitIdx,
                                             			rays->distance,
                                                        rays->alpha,
                                                        rays->beta),
                                             	SingleDistanceTriangleDepthFunctorWoop(geometry->vertices,
                                             										   geometry->bvhInnerNodes,
                                             										   geometry->bvhLeafNodes,
                                             										   INFINITE)),
                                             	"Intersect");
    eavlExecutor::Go();

 
    float maxDepth = 0;
    float minDepth = INFINITE;
    //for(int i=0; i< size; i++) if(rays->hitIdx->GetValue(i) != -1) cout<<"HIT";
    float acc = 0.f;
    for(int i=0; i< size; i++)
    {
        acc += rays->distance->GetValue(i);
        if( rays->distance->GetValue(i) == INFINITE) 
        {   
                rays->distance->SetValue(i,0);
        }
        else
        {

            maxDepth= max(rays->distance->GetValue(i), maxDepth);  
            minDepth= max(0.f,min(minDepth, rays->distance->GetValue(i)));//??
        }
        
    }
    cout<<"Total : "<<(int)acc<<endl; 
    cout<<"Depths "<<minDepth<<" "<<maxDepth<<endl;
    maxDepth = maxDepth - minDepth;

    for(int i = 0; i < size; i++) rays->distance->SetValue(i, (rays->distance->GetValue(i) - minDepth) / maxDepth);

    eavlExecutor::AddOperation(new_eavlScatterOp(eavlOpArgs(rays->distance),
                                                 eavlOpArgs(dummyFloat),
                                                 eavlOpArgs(cam->getPixelIndexes())),
                                                "scatter");
    eavlExecutor::Go();

    writeBMP(cam->getHeight(),cam->getWidth(),dummyFloat,dummyFloat,dummyFloat,"depth.bmp");

    delete dummyFloat;
    delete dummy;

}
